/*
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdint>

#define CUDA_SAFECALL(call)                                                 \
    {                                                                       \
        call;                                                               \
        cudaError err = cudaGetLastError();                                 \
        if (cudaSuccess != err) {                                           \
            fprintf(                                                        \
                stderr,                                                     \
                "Cuda error in function '%s' file '%s' in line %i : %s.\n", \
                #call, __FILE__, __LINE__, cudaGetErrorString(err));        \
            fflush(stderr);                                                 \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }


// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd_1(double *a, double *b, double *c, int n) {
    uint32_t start = clock();
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Make sure we do not go out of bounds
    if (id < n && a[id] > b[id]) c[id] = a[id] + b[id];
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd_2(double *a, double *b, double *c, int n) {
    uint32_t start = clock();
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n && a[id] <= b[id]) c[id] = a[id] + b[id];
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd_3(double *a, double *b, double *c, int n) {
    uint32_t start = clock();
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n && a[id] <= b[id]) c[id] = a[id] + b[id];
}



int main(int argc, char *argv[]) {
    // Size of vectors
    int n = 40960;
    if (argc > 1) n = atoi(argv[1]);

    // Host input vectors
    double *h_a_1;
    double *h_b_1;
    double *h_a_2;
    double *h_b_2;
    double *h_a_3;
    double *h_b_3;
    // Host output vector
    double *h_c_1;
    double *h_c_2;
    double *h_c_3;

    // Device input vectors
    double *d_a_1;
    double *d_b_1;
    double *d_a_2;
    double *d_b_2;
    double *d_a_3;
    double *d_b_3;
    // Device output vector
    double *d_c_1;
    double *d_c_2;
    double *d_c_3;

    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(double);

    // Allocate memory for each vector on host
    h_a_1 = (double *)malloc(bytes);
    h_a_2 = (double *)malloc(bytes);
    h_a_3 = (double *)malloc(bytes);
    h_b_1 = (double *)malloc(bytes);
    h_b_2 = (double *)malloc(bytes);
    h_b_3 = (double *)malloc(bytes);
    h_c_1 = (double *)malloc(bytes);
    h_c_2 = (double *)malloc(bytes);
    h_c_3 = (double *)malloc(bytes);

    // Allocate memory for each vector on GPU
    cudaMalloc(&d_a_1, bytes);
    cudaMalloc(&d_b_1, bytes);
    cudaMalloc(&d_c_1, bytes);
    cudaMalloc(&d_a_2, bytes);
    cudaMalloc(&d_b_2, bytes);
    cudaMalloc(&d_c_2, bytes);
    cudaMalloc(&d_a_3, bytes);
    cudaMalloc(&d_b_3, bytes);
    cudaMalloc(&d_c_3, bytes);

    int i;
    // Initialize vectors on host
    // for (i = 0; i < n; i++) {
    //     h_a_1[i] = sin(i) * sin(i);
    //     h_b_1[i] = cos(i) * cos(i);
    //     h_c_1[i] = 0;
    // }
    for (i = 0; i < n; i++) {
        h_a_1[i] = 0.1*i;
        h_b_1[i] = 0.1*i;
        h_c_1[i] = 0;
        h_a_2[i] = 0.1*i;
        h_b_2[i] = 0.1*i;
        h_c_2[i] = 0;
        h_a_3[i] = 0.1*i;
        h_b_3[i] = 0.1*i;
        h_c_3[i] = 0;
    }

    // Copy host vectors to device
    cudaMemcpy(d_a_1, h_a_1, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_b_1, h_b_1, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_c_1, h_c_1, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_a_2, h_a_2, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_b_2, h_b_2, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_c_2, h_c_2, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_a_3, h_a_3, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_b_3, h_b_3, bytes, cudaMemcpyHostToDevice);
    cudaMemcpy(d_c_3, h_c_3, bytes, cudaMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 256;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n / blockSize);
    
    // Execute the kernel
    CUDA_SAFECALL((vecAdd_1<<<gridSize, blockSize>>>(d_a_1, d_b_1, d_c_1, n)));
    CUDA_SAFECALL((vecAdd_2<<<gridSize, blockSize>>>(d_a_2, d_b_2, d_c_2, n)));
    CUDA_SAFECALL((vecAdd_3<<<gridSize, blockSize>>>(d_a_3, d_b_3, d_c_3, n)));
    
    // Copy array back to host
    cudaMemcpy(h_c_1, d_c_1, bytes, cudaMemcpyDeviceToHost);
    cudaMemcpy(h_c_2, d_c_2, bytes, cudaMemcpyDeviceToHost);
    cudaMemcpy(h_c_3, d_c_3, bytes, cudaMemcpyDeviceToHost);

    // Sum up vector c and print result divided by n, this should equal 1 within
    // error
    double sum;
    sum = 0;
    for (i = 0; i < n; i++) sum += h_c_1[i];
    printf("Final sum_1 = %f; sum_1/n = %f (should be ~1)\n", sum, sum / n);
    sum = 0;
    for (i = 0; i < n; i++) sum += h_c_2[i];
    printf("Final sum_2 = %f; sum_2/n = %f (should be ~1)\n", sum, sum / n);
    sum = 0;
    for (i = 0; i < n; i++) sum += h_c_3[i];
    printf("Final sum_3 = %f; sum_3/n = %f (should be ~1)\n", sum, sum / n);

    // Release device memory
    cudaFree(d_a_1);
    cudaFree(d_b_1);
    cudaFree(d_c_1);
    cudaFree(d_a_2);
    cudaFree(d_b_2);
    cudaFree(d_c_2);
    cudaFree(d_a_3);
    cudaFree(d_b_3);
    cudaFree(d_c_3);

    // Release host memory
    free(h_a_1);
    free(h_b_1);
    free(h_c_1);
    free(h_a_2);
    free(h_b_2);
    free(h_c_2);
    free(h_a_3);
    free(h_b_3);
    free(h_c_3);

    return 0;
}
*/

/**/
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <functional>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>

#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/library_types.h>

// CUDA API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

// cublas API error checking
#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("cublas error");                                              \
        }                                                                                          \
    } while (0)

// Returns cudaDataType value as defined in library_types.h for the string
// containing type name
hipDataType get_cuda_library_type(std::string type_string) {
    if (type_string.compare("CUDA_R_16F") == 0)
        return HIP_R_16F;
    else if (type_string.compare("CUDA_C_16F") == 0)
        return HIP_C_16F;
    else if (type_string.compare("CUDA_R_32F") == 0)
        return HIP_R_32F;
    else if (type_string.compare("CUDA_C_32F") == 0)
        return HIP_C_32F;
    else if (type_string.compare("CUDA_R_64F") == 0)
        return HIP_R_64F;
    else if (type_string.compare("CUDA_C_64F") == 0)
        return HIP_C_64F;
    else if (type_string.compare("CUDA_R_8I") == 0)
        return HIP_R_8I;
    else if (type_string.compare("CUDA_C_8I") == 0)
        return HIP_C_8I;
    else if (type_string.compare("CUDA_R_8U") == 0)
        return HIP_R_8U;
    else if (type_string.compare("CUDA_C_8U") == 0)
        return HIP_C_8U;
    else if (type_string.compare("CUDA_R_32I") == 0)
        return HIP_R_32I;
    else if (type_string.compare("CUDA_C_32I") == 0)
        return HIP_C_32I;
    else if (type_string.compare("CUDA_R_32U") == 0)
        return HIP_R_32U;
    else if (type_string.compare("CUDA_C_32U") == 0)
        return HIP_C_32U;
    else
        throw std::runtime_error("Unknown CUDA datatype");
}

__global__ void cublasHgemm111(                       hipblasHandle_t handle, 
                                                      hipblasOperation_t transa,
                                                      hipblasOperation_t transb, 
                                                      int m,
                                                      int n,
                                                      int k,
                                                      const __half *alpha, /* host or device pointer */  
                                                      const __half *A, 
                                                      int lda,
                                                      const __half *B,
                                                      int ldb, 
                                                      const __half *beta, /* host or device pointer */  
                                                      __half *C,
                                                      int ldc) {
    uint32_t start = clock();
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    C[id] = A[id];
}

using data_type = half;

int main(int argc, char *argv[]) {
    
    int m = 257;
    int k = 1;
    int n = 513; 
    // int m = 513;
    // int k = 513;
    // int n = 513; 
    
    const int lda = k;
    const int ldb = n;
    const int ldc = n;

    hipblasHandle_t cublasH1,cublasH2,cublasH3,cublasH4,cublasH5,cublasH6,cublasH7,cublasH8,cublasH9,cublasH10,cublasH11,cublasH12,cublasH13,cublasH14,cublasH15,cublasH16,cublasH17,cublasH18,cublasH19,cublasH20,cublasH21,cublasH22,cublasH23,cublasH24,cublasH25,cublasH26,cublasH27,cublasH28,cublasH29,cublasH30,cublasH31,cublasH32,cublasH33,cublasH34,cublasH35,cublasH36,cublasH37,cublasH38,cublasH39,cublasH40,cublasH41,cublasH42,cublasH43,cublasH44,cublasH45,cublasH46,cublasH47,cublasH48,cublasH49,cublasH50,cublasH51,cublasH52,cublasH53,cublasH54,cublasH55,cublasH56,cublasH57,cublasH58,cublasH59,cublasH60,cublasH61,cublasH62,cublasH63,cublasH64,cublasH65,cublasH66,cublasH67,cublasH68,cublasH69,cublasH70,cublasH71,cublasH72,cublasH73,cublasH74,cublasH75,cublasH76,cublasH77,cublasH78,cublasH79,cublasH80,cublasH81 = NULL;


    hipStream_t stream1,stream2,stream3,stream4,stream5,stream6,stream7,stream8,stream9,stream10,stream11,stream12,stream13,stream14,stream15,stream16,stream17,stream18,stream19,stream20,stream21,stream22,stream23,stream24,stream25,stream26,stream27,stream28,stream29,stream30,stream31,stream32,stream33,stream34,stream35,stream36,stream37,stream38,stream39,stream40,stream41,stream42,stream43,stream44,stream45,stream46,stream47,stream48,stream49,stream50,stream51,stream52,stream53,stream54,stream55,stream56,stream57,stream58,stream59,stream60,stream61,stream62,stream63,stream64,stream65,stream66,stream67,stream68,stream69,stream70,stream71,stream72,stream73,stream74,stream75,stream76,stream77,stream78,stream79,stream80,stream81 = NULL;

    // std::random_device rd;
    // std::mt19937 gen(rd());
    // std::uniform_real_distribution<> dis(0.2, 1.0);

    //
    //   A = | 1.0 | 2.0 |
    //       | 3.0 | 4.0 |
    //
    //   B = | 5.0 | 6.0 |
    //       | 7.0 | 8.0 |
    //

    // const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    // const std::vector<data_type> B = {5.0, 6.0, 7.0, 8.0};
    std::vector<data_type> A(m * k, 1./13.);
    std::vector<data_type> B(k * n, 1./13.);

    // for(std::vector<data_type>::iterator it = A.begin(); it != A.end(); ++it)
    //     *it = dis(gen);
    // for(std::vector<data_type>::iterator it = B.begin(); it != B.end(); ++it)
    //     *it = dis(gen);

    std::vector<data_type> C(m * n);
    const data_type alpha = 1.0;
    const data_type beta = 0.0;

    // define 71 d_a_x device data
    data_type *d_a_1 = nullptr;
    data_type *d_b_1 = nullptr;
    data_type *d_c_1 = nullptr;
    data_type *d_a_2 = nullptr;
    data_type *d_b_2 = nullptr;
    data_type *d_c_2 = nullptr;
    data_type *d_a_3 = nullptr;
    data_type *d_b_3 = nullptr;
    data_type *d_c_3 = nullptr;
    data_type *d_a_4 = nullptr;
    data_type *d_b_4 = nullptr;
    data_type *d_c_4 = nullptr;
    data_type *d_a_5 = nullptr;
    data_type *d_b_5 = nullptr;
    data_type *d_c_5 = nullptr;
    data_type *d_a_6 = nullptr;
    data_type *d_b_6 = nullptr;
    data_type *d_c_6 = nullptr;
    data_type *d_a_7 = nullptr;
    data_type *d_b_7 = nullptr;
    data_type *d_c_7 = nullptr;
    data_type *d_a_8 = nullptr;
    data_type *d_b_8 = nullptr;
    data_type *d_c_8 = nullptr;
    data_type *d_a_9 = nullptr;
    data_type *d_b_9 = nullptr;
    data_type *d_c_9 = nullptr;
    data_type *d_a_10 = nullptr;
    data_type *d_b_10 = nullptr;
    data_type *d_c_10 = nullptr;
    data_type *d_a_11 = nullptr;
    data_type *d_b_11 = nullptr;
    data_type *d_c_11 = nullptr;
    data_type *d_a_12 = nullptr;
    data_type *d_b_12 = nullptr;
    data_type *d_c_12 = nullptr;
    data_type *d_a_13 = nullptr;
    data_type *d_b_13 = nullptr;
    data_type *d_c_13 = nullptr;
    data_type *d_a_14 = nullptr;
    data_type *d_b_14 = nullptr;
    data_type *d_c_14 = nullptr;
    data_type *d_a_15 = nullptr;
    data_type *d_b_15 = nullptr;
    data_type *d_c_15 = nullptr;
    data_type *d_a_16 = nullptr;
    data_type *d_b_16 = nullptr;
    data_type *d_c_16 = nullptr;
    data_type *d_a_17 = nullptr;
    data_type *d_b_17 = nullptr;
    data_type *d_c_17 = nullptr;
    data_type *d_a_18 = nullptr;
    data_type *d_b_18 = nullptr;
    data_type *d_c_18 = nullptr;
    data_type *d_a_19 = nullptr;
    data_type *d_b_19 = nullptr;
    data_type *d_c_19 = nullptr;
    data_type *d_a_20 = nullptr;
    data_type *d_b_20 = nullptr;
    data_type *d_c_20 = nullptr;
    data_type *d_a_21 = nullptr;
    data_type *d_b_21 = nullptr;
    data_type *d_c_21 = nullptr;
    data_type *d_a_22 = nullptr;
    data_type *d_b_22 = nullptr;
    data_type *d_c_22 = nullptr;
    data_type *d_a_23 = nullptr;
    data_type *d_b_23 = nullptr;
    data_type *d_c_23 = nullptr;
    data_type *d_a_24 = nullptr;
    data_type *d_b_24 = nullptr;
    data_type *d_c_24 = nullptr;
    data_type *d_a_25 = nullptr;
    data_type *d_b_25 = nullptr;
    data_type *d_c_25 = nullptr;
    data_type *d_a_26 = nullptr;
    data_type *d_b_26 = nullptr;
    data_type *d_c_26 = nullptr;
    data_type *d_a_27 = nullptr;
    data_type *d_b_27 = nullptr;
    data_type *d_c_27 = nullptr;
    data_type *d_a_28 = nullptr;
    data_type *d_b_28 = nullptr;
    data_type *d_c_28 = nullptr;
    data_type *d_a_29 = nullptr;
    data_type *d_b_29 = nullptr;
    data_type *d_c_29 = nullptr;
    data_type *d_a_30 = nullptr;
    data_type *d_b_30 = nullptr;
    data_type *d_c_30 = nullptr;
    data_type *d_a_31 = nullptr;
    data_type *d_b_31 = nullptr;
    data_type *d_c_31 = nullptr;
    data_type *d_a_32 = nullptr;
    data_type *d_b_32 = nullptr;
    data_type *d_c_32 = nullptr;
    data_type *d_a_33 = nullptr;
    data_type *d_b_33 = nullptr;
    data_type *d_c_33 = nullptr;
    data_type *d_a_34 = nullptr;
    data_type *d_b_34 = nullptr;
    data_type *d_c_34 = nullptr;
    data_type *d_a_35 = nullptr;
    data_type *d_b_35 = nullptr;
    data_type *d_c_35 = nullptr;
    data_type *d_a_36 = nullptr;
    data_type *d_b_36 = nullptr;
    data_type *d_c_36 = nullptr;
    data_type *d_a_37 = nullptr;
    data_type *d_b_37 = nullptr;
    data_type *d_c_37 = nullptr;
    data_type *d_a_38 = nullptr;
    data_type *d_b_38 = nullptr;
    data_type *d_c_38 = nullptr;
    data_type *d_a_39 = nullptr;
    data_type *d_b_39 = nullptr;
    data_type *d_c_39 = nullptr;
    data_type *d_a_40 = nullptr;
    data_type *d_b_40 = nullptr;
    data_type *d_c_40 = nullptr;
    data_type *d_a_41 = nullptr;
    data_type *d_b_41 = nullptr;
    data_type *d_c_41 = nullptr;
    data_type *d_a_42 = nullptr;
    data_type *d_b_42 = nullptr;
    data_type *d_c_42 = nullptr;
    data_type *d_a_43 = nullptr;
    data_type *d_b_43 = nullptr;
    data_type *d_c_43 = nullptr;
    data_type *d_a_44 = nullptr;
    data_type *d_b_44 = nullptr;
    data_type *d_c_44 = nullptr;
    data_type *d_a_45 = nullptr;
    data_type *d_b_45 = nullptr;
    data_type *d_c_45 = nullptr;
    data_type *d_a_46 = nullptr;
    data_type *d_b_46 = nullptr;
    data_type *d_c_46 = nullptr;
    data_type *d_a_47 = nullptr;
    data_type *d_b_47 = nullptr;
    data_type *d_c_47 = nullptr;
    data_type *d_a_48 = nullptr;
    data_type *d_b_48 = nullptr;
    data_type *d_c_48 = nullptr;
    data_type *d_a_49 = nullptr;
    data_type *d_b_49 = nullptr;
    data_type *d_c_49 = nullptr;
    data_type *d_a_50 = nullptr;
    data_type *d_b_50 = nullptr;
    data_type *d_c_50 = nullptr;
    data_type *d_a_51 = nullptr;
    data_type *d_b_51 = nullptr;
    data_type *d_c_51 = nullptr;
    data_type *d_a_52 = nullptr;
    data_type *d_b_52 = nullptr;
    data_type *d_c_52 = nullptr;
    data_type *d_a_53 = nullptr;
    data_type *d_b_53 = nullptr;
    data_type *d_c_53 = nullptr;
    data_type *d_a_54 = nullptr;
    data_type *d_b_54 = nullptr;
    data_type *d_c_54 = nullptr;
    data_type *d_a_55 = nullptr;
    data_type *d_b_55 = nullptr;
    data_type *d_c_55 = nullptr;
    data_type *d_a_56 = nullptr;
    data_type *d_b_56 = nullptr;
    data_type *d_c_56 = nullptr;
    data_type *d_a_57 = nullptr;
    data_type *d_b_57 = nullptr;
    data_type *d_c_57 = nullptr;
    data_type *d_a_58 = nullptr;
    data_type *d_b_58 = nullptr;
    data_type *d_c_58 = nullptr;
    data_type *d_a_59 = nullptr;
    data_type *d_b_59 = nullptr;
    data_type *d_c_59 = nullptr;
    data_type *d_a_60 = nullptr;
    data_type *d_b_60 = nullptr;
    data_type *d_c_60 = nullptr;
    data_type *d_a_61 = nullptr;
    data_type *d_b_61 = nullptr;
    data_type *d_c_61 = nullptr;
    data_type *d_a_62 = nullptr;
    data_type *d_b_62 = nullptr;
    data_type *d_c_62 = nullptr;
    data_type *d_a_63 = nullptr;
    data_type *d_b_63 = nullptr;
    data_type *d_c_63 = nullptr;
    data_type *d_a_64 = nullptr;
    data_type *d_b_64 = nullptr;
    data_type *d_c_64 = nullptr;
    data_type *d_a_65 = nullptr;
    data_type *d_b_65 = nullptr;
    data_type *d_c_65 = nullptr;
    data_type *d_a_66 = nullptr;
    data_type *d_b_66 = nullptr;
    data_type *d_c_66 = nullptr;
    data_type *d_a_67 = nullptr;
    data_type *d_b_67 = nullptr;
    data_type *d_c_67 = nullptr;
    data_type *d_a_68 = nullptr;
    data_type *d_b_68 = nullptr;
    data_type *d_c_68 = nullptr;
    data_type *d_a_69 = nullptr;
    data_type *d_b_69 = nullptr;
    data_type *d_c_69 = nullptr;
    data_type *d_a_70 = nullptr;
    data_type *d_b_70 = nullptr;
    data_type *d_c_70 = nullptr;
    data_type *d_a_71 = nullptr;
    data_type *d_b_71 = nullptr;
    data_type *d_c_71 = nullptr;
    data_type *d_a_72 = nullptr;
    data_type *d_b_72 = nullptr;
    data_type *d_c_72 = nullptr;
    data_type *d_a_73 = nullptr;
    data_type *d_b_73 = nullptr;
    data_type *d_c_73 = nullptr;
    data_type *d_a_74 = nullptr;
    data_type *d_b_74 = nullptr;
    data_type *d_c_74 = nullptr;
    data_type *d_a_75 = nullptr;
    data_type *d_b_75 = nullptr;
    data_type *d_c_75 = nullptr;
    data_type *d_a_76 = nullptr;
    data_type *d_b_76 = nullptr;
    data_type *d_c_76 = nullptr;
    data_type *d_a_77 = nullptr;
    data_type *d_b_77 = nullptr;
    data_type *d_c_77 = nullptr;
    data_type *d_a_78 = nullptr;
    data_type *d_b_78 = nullptr;
    data_type *d_c_78 = nullptr;
    data_type *d_a_79 = nullptr;
    data_type *d_b_79 = nullptr;
    data_type *d_c_79 = nullptr;
    data_type *d_a_80 = nullptr;
    data_type *d_b_80 = nullptr;
    data_type *d_c_80 = nullptr;
    data_type *d_a_81 = nullptr;
    data_type *d_b_81 = nullptr;
    data_type *d_c_81 = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;


    // step 1: create cublas handle, bind a stream 
    CUBLAS_CHECK(hipblasCreate(&cublasH1));CUBLAS_CHECK(hipblasCreate(&cublasH2));CUBLAS_CHECK(hipblasCreate(&cublasH3));CUBLAS_CHECK(hipblasCreate(&cublasH4));CUBLAS_CHECK(hipblasCreate(&cublasH5));CUBLAS_CHECK(hipblasCreate(&cublasH6));CUBLAS_CHECK(hipblasCreate(&cublasH7));CUBLAS_CHECK(hipblasCreate(&cublasH8));CUBLAS_CHECK(hipblasCreate(&cublasH9));CUBLAS_CHECK(hipblasCreate(&cublasH10));CUBLAS_CHECK(hipblasCreate(&cublasH11));CUBLAS_CHECK(hipblasCreate(&cublasH12));CUBLAS_CHECK(hipblasCreate(&cublasH13));CUBLAS_CHECK(hipblasCreate(&cublasH14));CUBLAS_CHECK(hipblasCreate(&cublasH15));CUBLAS_CHECK(hipblasCreate(&cublasH16));CUBLAS_CHECK(hipblasCreate(&cublasH17));CUBLAS_CHECK(hipblasCreate(&cublasH18));CUBLAS_CHECK(hipblasCreate(&cublasH19));CUBLAS_CHECK(hipblasCreate(&cublasH20));CUBLAS_CHECK(hipblasCreate(&cublasH21));CUBLAS_CHECK(hipblasCreate(&cublasH22));CUBLAS_CHECK(hipblasCreate(&cublasH23));CUBLAS_CHECK(hipblasCreate(&cublasH24));CUBLAS_CHECK(hipblasCreate(&cublasH25));CUBLAS_CHECK(hipblasCreate(&cublasH26));CUBLAS_CHECK(hipblasCreate(&cublasH27));CUBLAS_CHECK(hipblasCreate(&cublasH28));CUBLAS_CHECK(hipblasCreate(&cublasH29));CUBLAS_CHECK(hipblasCreate(&cublasH30));CUBLAS_CHECK(hipblasCreate(&cublasH31));CUBLAS_CHECK(hipblasCreate(&cublasH32));CUBLAS_CHECK(hipblasCreate(&cublasH33));CUBLAS_CHECK(hipblasCreate(&cublasH34));CUBLAS_CHECK(hipblasCreate(&cublasH35));CUBLAS_CHECK(hipblasCreate(&cublasH36));CUBLAS_CHECK(hipblasCreate(&cublasH37));CUBLAS_CHECK(hipblasCreate(&cublasH38));CUBLAS_CHECK(hipblasCreate(&cublasH39));CUBLAS_CHECK(hipblasCreate(&cublasH40));CUBLAS_CHECK(hipblasCreate(&cublasH41));CUBLAS_CHECK(hipblasCreate(&cublasH42));CUBLAS_CHECK(hipblasCreate(&cublasH43));CUBLAS_CHECK(hipblasCreate(&cublasH44));CUBLAS_CHECK(hipblasCreate(&cublasH45));CUBLAS_CHECK(hipblasCreate(&cublasH46));CUBLAS_CHECK(hipblasCreate(&cublasH47));CUBLAS_CHECK(hipblasCreate(&cublasH48));CUBLAS_CHECK(hipblasCreate(&cublasH49));CUBLAS_CHECK(hipblasCreate(&cublasH50));CUBLAS_CHECK(hipblasCreate(&cublasH51));CUBLAS_CHECK(hipblasCreate(&cublasH52));CUBLAS_CHECK(hipblasCreate(&cublasH53));CUBLAS_CHECK(hipblasCreate(&cublasH54));CUBLAS_CHECK(hipblasCreate(&cublasH55));CUBLAS_CHECK(hipblasCreate(&cublasH56));CUBLAS_CHECK(hipblasCreate(&cublasH57));CUBLAS_CHECK(hipblasCreate(&cublasH58));CUBLAS_CHECK(hipblasCreate(&cublasH59));CUBLAS_CHECK(hipblasCreate(&cublasH60));CUBLAS_CHECK(hipblasCreate(&cublasH61));CUBLAS_CHECK(hipblasCreate(&cublasH62));CUBLAS_CHECK(hipblasCreate(&cublasH63));CUBLAS_CHECK(hipblasCreate(&cublasH64));CUBLAS_CHECK(hipblasCreate(&cublasH65));CUBLAS_CHECK(hipblasCreate(&cublasH66));CUBLAS_CHECK(hipblasCreate(&cublasH67));CUBLAS_CHECK(hipblasCreate(&cublasH68));CUBLAS_CHECK(hipblasCreate(&cublasH69));CUBLAS_CHECK(hipblasCreate(&cublasH70));CUBLAS_CHECK(hipblasCreate(&cublasH71));CUBLAS_CHECK(hipblasCreate(&cublasH72));CUBLAS_CHECK(hipblasCreate(&cublasH73));CUBLAS_CHECK(hipblasCreate(&cublasH74));CUBLAS_CHECK(hipblasCreate(&cublasH75));CUBLAS_CHECK(hipblasCreate(&cublasH76));CUBLAS_CHECK(hipblasCreate(&cublasH77));CUBLAS_CHECK(hipblasCreate(&cublasH78));CUBLAS_CHECK(hipblasCreate(&cublasH79));CUBLAS_CHECK(hipblasCreate(&cublasH80));CUBLAS_CHECK(hipblasCreate(&cublasH81));
    

    CUDA_CHECK(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream4, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream5, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream6, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream7, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream8, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream9, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream10, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream11, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream12, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream13, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream14, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream15, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream16, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream17, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream18, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream19, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream20, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream21, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream22, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream23, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream24, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream25, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream26, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream27, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream28, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream29, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream30, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream31, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream32, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream33, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream34, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream35, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream36, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream37, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream38, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream39, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream40, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream41, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream42, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream43, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream44, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream45, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream46, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream47, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream48, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream49, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream50, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream51, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream52, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream53, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream54, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream55, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream56, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream57, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream58, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream59, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream60, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream61, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream62, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream63, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream64, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream65, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream66, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream67, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream68, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream69, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream70, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream71, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream72, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream73, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream74, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream75, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream76, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream77, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream78, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream79, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream80, hipStreamNonBlocking));CUDA_CHECK(hipStreamCreateWithFlags(&stream81, hipStreamNonBlocking));


    CUBLAS_CHECK(hipblasSetStream(cublasH1, stream1));CUBLAS_CHECK(hipblasSetStream(cublasH2, stream2));CUBLAS_CHECK(hipblasSetStream(cublasH3, stream3));CUBLAS_CHECK(hipblasSetStream(cublasH4, stream4));CUBLAS_CHECK(hipblasSetStream(cublasH5, stream5));CUBLAS_CHECK(hipblasSetStream(cublasH6, stream6));CUBLAS_CHECK(hipblasSetStream(cublasH7, stream7));CUBLAS_CHECK(hipblasSetStream(cublasH8, stream8));CUBLAS_CHECK(hipblasSetStream(cublasH9, stream9));CUBLAS_CHECK(hipblasSetStream(cublasH10, stream10));CUBLAS_CHECK(hipblasSetStream(cublasH11, stream11));CUBLAS_CHECK(hipblasSetStream(cublasH12, stream12));CUBLAS_CHECK(hipblasSetStream(cublasH13, stream13));CUBLAS_CHECK(hipblasSetStream(cublasH14, stream14));CUBLAS_CHECK(hipblasSetStream(cublasH15, stream15));CUBLAS_CHECK(hipblasSetStream(cublasH16, stream16));CUBLAS_CHECK(hipblasSetStream(cublasH17, stream17));CUBLAS_CHECK(hipblasSetStream(cublasH18, stream18));CUBLAS_CHECK(hipblasSetStream(cublasH19, stream19));CUBLAS_CHECK(hipblasSetStream(cublasH20, stream20));CUBLAS_CHECK(hipblasSetStream(cublasH21, stream21));CUBLAS_CHECK(hipblasSetStream(cublasH22, stream22));CUBLAS_CHECK(hipblasSetStream(cublasH23, stream23));CUBLAS_CHECK(hipblasSetStream(cublasH24, stream24));CUBLAS_CHECK(hipblasSetStream(cublasH25, stream25));CUBLAS_CHECK(hipblasSetStream(cublasH26, stream26));CUBLAS_CHECK(hipblasSetStream(cublasH27, stream27));CUBLAS_CHECK(hipblasSetStream(cublasH28, stream28));CUBLAS_CHECK(hipblasSetStream(cublasH29, stream29));CUBLAS_CHECK(hipblasSetStream(cublasH30, stream30));CUBLAS_CHECK(hipblasSetStream(cublasH31, stream31));CUBLAS_CHECK(hipblasSetStream(cublasH32, stream32));CUBLAS_CHECK(hipblasSetStream(cublasH33, stream33));CUBLAS_CHECK(hipblasSetStream(cublasH34, stream34));CUBLAS_CHECK(hipblasSetStream(cublasH35, stream35));CUBLAS_CHECK(hipblasSetStream(cublasH36, stream36));CUBLAS_CHECK(hipblasSetStream(cublasH37, stream37));CUBLAS_CHECK(hipblasSetStream(cublasH38, stream38));CUBLAS_CHECK(hipblasSetStream(cublasH39, stream39));CUBLAS_CHECK(hipblasSetStream(cublasH40, stream40));CUBLAS_CHECK(hipblasSetStream(cublasH41, stream41));CUBLAS_CHECK(hipblasSetStream(cublasH42, stream42));CUBLAS_CHECK(hipblasSetStream(cublasH43, stream43));CUBLAS_CHECK(hipblasSetStream(cublasH44, stream44));CUBLAS_CHECK(hipblasSetStream(cublasH45, stream45));CUBLAS_CHECK(hipblasSetStream(cublasH46, stream46));CUBLAS_CHECK(hipblasSetStream(cublasH47, stream47));CUBLAS_CHECK(hipblasSetStream(cublasH48, stream48));CUBLAS_CHECK(hipblasSetStream(cublasH49, stream49));CUBLAS_CHECK(hipblasSetStream(cublasH50, stream50));CUBLAS_CHECK(hipblasSetStream(cublasH51, stream51));CUBLAS_CHECK(hipblasSetStream(cublasH52, stream52));CUBLAS_CHECK(hipblasSetStream(cublasH53, stream53));CUBLAS_CHECK(hipblasSetStream(cublasH54, stream54));CUBLAS_CHECK(hipblasSetStream(cublasH55, stream55));CUBLAS_CHECK(hipblasSetStream(cublasH56, stream56));CUBLAS_CHECK(hipblasSetStream(cublasH57, stream57));CUBLAS_CHECK(hipblasSetStream(cublasH58, stream58));CUBLAS_CHECK(hipblasSetStream(cublasH59, stream59));CUBLAS_CHECK(hipblasSetStream(cublasH60, stream60));CUBLAS_CHECK(hipblasSetStream(cublasH61, stream61));CUBLAS_CHECK(hipblasSetStream(cublasH62, stream62));CUBLAS_CHECK(hipblasSetStream(cublasH63, stream63));CUBLAS_CHECK(hipblasSetStream(cublasH64, stream64));CUBLAS_CHECK(hipblasSetStream(cublasH65, stream65));CUBLAS_CHECK(hipblasSetStream(cublasH66, stream66));CUBLAS_CHECK(hipblasSetStream(cublasH67, stream67));CUBLAS_CHECK(hipblasSetStream(cublasH68, stream68));CUBLAS_CHECK(hipblasSetStream(cublasH69, stream69));CUBLAS_CHECK(hipblasSetStream(cublasH70, stream70));CUBLAS_CHECK(hipblasSetStream(cublasH71, stream71));CUBLAS_CHECK(hipblasSetStream(cublasH72, stream72));CUBLAS_CHECK(hipblasSetStream(cublasH73, stream73));CUBLAS_CHECK(hipblasSetStream(cublasH74, stream74));CUBLAS_CHECK(hipblasSetStream(cublasH75, stream75));CUBLAS_CHECK(hipblasSetStream(cublasH76, stream76));CUBLAS_CHECK(hipblasSetStream(cublasH77, stream77));CUBLAS_CHECK(hipblasSetStream(cublasH78, stream78));CUBLAS_CHECK(hipblasSetStream(cublasH79, stream79));CUBLAS_CHECK(hipblasSetStream(cublasH80, stream80));CUBLAS_CHECK(hipblasSetStream(cublasH81, stream81));

    // step 2: copy data to device 
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_1), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_1), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_1), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_2), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_2), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_2), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_3), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_3), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_3), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_4), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_4), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_4), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_5), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_5), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_5), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_6), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_6), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_6), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_7), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_7), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_7), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_8), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_8), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_8), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_9), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_9), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_9), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_10), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_10), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_10), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_11), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_11), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_11), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_12), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_12), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_12), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_13), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_13), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_13), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_14), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_14), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_14), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_15), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_15), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_15), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_16), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_16), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_16), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_17), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_17), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_17), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_18), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_18), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_18), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_19), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_19), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_19), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_20), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_20), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_20), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_21), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_21), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_21), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_22), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_22), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_22), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_23), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_23), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_23), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_24), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_24), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_24), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_25), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_25), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_25), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_26), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_26), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_26), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_27), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_27), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_27), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_28), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_28), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_28), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_29), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_29), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_29), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_30), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_30), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_30), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_31), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_31), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_31), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_32), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_32), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_32), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_33), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_33), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_33), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_34), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_34), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_34), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_35), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_35), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_35), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_36), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_36), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_36), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_37), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_37), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_37), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_38), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_38), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_38), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_39), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_39), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_39), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_40), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_40), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_40), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_41), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_41), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_41), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_42), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_42), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_42), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_43), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_43), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_43), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_44), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_44), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_44), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_45), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_45), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_45), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_46), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_46), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_46), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_47), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_47), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_47), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_48), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_48), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_48), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_49), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_49), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_49), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_50), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_50), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_50), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_51), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_51), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_51), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_52), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_52), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_52), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_53), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_53), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_53), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_54), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_54), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_54), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_55), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_55), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_55), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_56), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_56), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_56), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_57), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_57), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_57), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_58), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_58), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_58), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_59), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_59), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_59), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_60), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_60), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_60), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_61), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_61), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_61), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_62), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_62), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_62), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_63), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_63), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_63), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_64), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_64), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_64), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_65), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_65), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_65), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_66), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_66), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_66), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_67), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_67), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_67), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_68), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_68), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_68), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_69), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_69), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_69), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_70), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_70), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_70), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_71), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_71), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_71), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_72), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_72), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_72), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_73), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_73), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_73), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_74), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_74), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_74), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_75), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_75), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_75), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_76), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_76), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_76), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_77), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_77), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_77), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_78), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_78), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_78), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_79), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_79), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_79), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_80), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_80), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_80), sizeof(data_type) * C.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_a_81), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b_81), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_c_81), sizeof(data_type) * C.size()));

    CUDA_CHECK(hipMemcpyAsync(d_a_1, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream1));
    CUDA_CHECK(hipMemcpyAsync(d_b_1, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream1));
    CUDA_CHECK(hipMemcpyAsync(d_a_2, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream2));
    CUDA_CHECK(hipMemcpyAsync(d_b_2, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream2));
    CUDA_CHECK(hipMemcpyAsync(d_a_3, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream3));
    CUDA_CHECK(hipMemcpyAsync(d_b_3, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream3));
    CUDA_CHECK(hipMemcpyAsync(d_a_4, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream4));
    CUDA_CHECK(hipMemcpyAsync(d_b_4, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream4));
    CUDA_CHECK(hipMemcpyAsync(d_a_5, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream5));
    CUDA_CHECK(hipMemcpyAsync(d_b_5, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream5));
    CUDA_CHECK(hipMemcpyAsync(d_a_6, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream6));
    CUDA_CHECK(hipMemcpyAsync(d_b_6, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream6));
    CUDA_CHECK(hipMemcpyAsync(d_a_7, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream7));
    CUDA_CHECK(hipMemcpyAsync(d_b_7, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream7));
    CUDA_CHECK(hipMemcpyAsync(d_a_8, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream8));
    CUDA_CHECK(hipMemcpyAsync(d_b_8, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream8));
    CUDA_CHECK(hipMemcpyAsync(d_a_9, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream9));
    CUDA_CHECK(hipMemcpyAsync(d_b_9, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream9));
    CUDA_CHECK(hipMemcpyAsync(d_a_10, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream10));
    CUDA_CHECK(hipMemcpyAsync(d_b_10, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream10));
    CUDA_CHECK(hipMemcpyAsync(d_a_11, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream11));
    CUDA_CHECK(hipMemcpyAsync(d_b_11, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream11));
    CUDA_CHECK(hipMemcpyAsync(d_a_12, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream12));
    CUDA_CHECK(hipMemcpyAsync(d_b_12, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream12));
    CUDA_CHECK(hipMemcpyAsync(d_a_13, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream13));
    CUDA_CHECK(hipMemcpyAsync(d_b_13, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream13));
    CUDA_CHECK(hipMemcpyAsync(d_a_14, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream14));
    CUDA_CHECK(hipMemcpyAsync(d_b_14, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream14));
    CUDA_CHECK(hipMemcpyAsync(d_a_15, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream15));
    CUDA_CHECK(hipMemcpyAsync(d_b_15, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream15));
    CUDA_CHECK(hipMemcpyAsync(d_a_16, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream16));
    CUDA_CHECK(hipMemcpyAsync(d_b_16, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream16));
    CUDA_CHECK(hipMemcpyAsync(d_a_17, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream17));
    CUDA_CHECK(hipMemcpyAsync(d_b_17, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream17));
    CUDA_CHECK(hipMemcpyAsync(d_a_18, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream18));
    CUDA_CHECK(hipMemcpyAsync(d_b_18, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream18));
    CUDA_CHECK(hipMemcpyAsync(d_a_19, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream19));
    CUDA_CHECK(hipMemcpyAsync(d_b_19, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream19));
    CUDA_CHECK(hipMemcpyAsync(d_a_20, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream20));
    CUDA_CHECK(hipMemcpyAsync(d_b_20, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream20));
    CUDA_CHECK(hipMemcpyAsync(d_a_21, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream21));
    CUDA_CHECK(hipMemcpyAsync(d_b_21, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream21));
    CUDA_CHECK(hipMemcpyAsync(d_a_22, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream22));
    CUDA_CHECK(hipMemcpyAsync(d_b_22, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream22));
    CUDA_CHECK(hipMemcpyAsync(d_a_23, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream23));
    CUDA_CHECK(hipMemcpyAsync(d_b_23, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream23));
    CUDA_CHECK(hipMemcpyAsync(d_a_24, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream24));
    CUDA_CHECK(hipMemcpyAsync(d_b_24, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream24));
    CUDA_CHECK(hipMemcpyAsync(d_a_25, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream25));
    CUDA_CHECK(hipMemcpyAsync(d_b_25, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream25));
    CUDA_CHECK(hipMemcpyAsync(d_a_26, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream26));
    CUDA_CHECK(hipMemcpyAsync(d_b_26, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream26));
    CUDA_CHECK(hipMemcpyAsync(d_a_27, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream27));
    CUDA_CHECK(hipMemcpyAsync(d_b_27, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream27));
    CUDA_CHECK(hipMemcpyAsync(d_a_28, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream28));
    CUDA_CHECK(hipMemcpyAsync(d_b_28, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream28));
    CUDA_CHECK(hipMemcpyAsync(d_a_29, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream29));
    CUDA_CHECK(hipMemcpyAsync(d_b_29, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream29));
    CUDA_CHECK(hipMemcpyAsync(d_a_30, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream30));
    CUDA_CHECK(hipMemcpyAsync(d_b_30, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream30));
    CUDA_CHECK(hipMemcpyAsync(d_a_31, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream31));
    CUDA_CHECK(hipMemcpyAsync(d_b_31, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream31));
    CUDA_CHECK(hipMemcpyAsync(d_a_32, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream32));
    CUDA_CHECK(hipMemcpyAsync(d_b_32, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream32));
    CUDA_CHECK(hipMemcpyAsync(d_a_33, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream33));
    CUDA_CHECK(hipMemcpyAsync(d_b_33, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream33));
    CUDA_CHECK(hipMemcpyAsync(d_a_34, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream34));
    CUDA_CHECK(hipMemcpyAsync(d_b_34, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream34));
    CUDA_CHECK(hipMemcpyAsync(d_a_35, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream35));
    CUDA_CHECK(hipMemcpyAsync(d_b_35, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream35));
    CUDA_CHECK(hipMemcpyAsync(d_a_36, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream36));
    CUDA_CHECK(hipMemcpyAsync(d_b_36, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream36));
    CUDA_CHECK(hipMemcpyAsync(d_a_37, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream37));
    CUDA_CHECK(hipMemcpyAsync(d_b_37, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream37));
    CUDA_CHECK(hipMemcpyAsync(d_a_38, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream38));
    CUDA_CHECK(hipMemcpyAsync(d_b_38, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream38));
    CUDA_CHECK(hipMemcpyAsync(d_a_39, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream39));
    CUDA_CHECK(hipMemcpyAsync(d_b_39, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream39));
    CUDA_CHECK(hipMemcpyAsync(d_a_40, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream40));
    CUDA_CHECK(hipMemcpyAsync(d_b_40, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream40));
    CUDA_CHECK(hipMemcpyAsync(d_a_41, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream41));
    CUDA_CHECK(hipMemcpyAsync(d_b_41, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream41));
    CUDA_CHECK(hipMemcpyAsync(d_a_42, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream42));
    CUDA_CHECK(hipMemcpyAsync(d_b_42, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream42));
    CUDA_CHECK(hipMemcpyAsync(d_a_43, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream43));
    CUDA_CHECK(hipMemcpyAsync(d_b_43, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream43));
    CUDA_CHECK(hipMemcpyAsync(d_a_44, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream44));
    CUDA_CHECK(hipMemcpyAsync(d_b_44, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream44));
    CUDA_CHECK(hipMemcpyAsync(d_a_45, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream45));
    CUDA_CHECK(hipMemcpyAsync(d_b_45, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream45));
    CUDA_CHECK(hipMemcpyAsync(d_a_46, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream46));
    CUDA_CHECK(hipMemcpyAsync(d_b_46, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream46));
    CUDA_CHECK(hipMemcpyAsync(d_a_47, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream47));
    CUDA_CHECK(hipMemcpyAsync(d_b_47, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream47));
    CUDA_CHECK(hipMemcpyAsync(d_a_48, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream48));
    CUDA_CHECK(hipMemcpyAsync(d_b_48, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream48));
    CUDA_CHECK(hipMemcpyAsync(d_a_49, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream49));
    CUDA_CHECK(hipMemcpyAsync(d_b_49, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream49));
    CUDA_CHECK(hipMemcpyAsync(d_a_50, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream50));
    CUDA_CHECK(hipMemcpyAsync(d_b_50, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream50));
    CUDA_CHECK(hipMemcpyAsync(d_a_51, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream51));
    CUDA_CHECK(hipMemcpyAsync(d_b_51, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream51));
    CUDA_CHECK(hipMemcpyAsync(d_a_52, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream52));
    CUDA_CHECK(hipMemcpyAsync(d_b_52, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream52));
    CUDA_CHECK(hipMemcpyAsync(d_a_53, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream53));
    CUDA_CHECK(hipMemcpyAsync(d_b_53, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream53));
    CUDA_CHECK(hipMemcpyAsync(d_a_54, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream54));
    CUDA_CHECK(hipMemcpyAsync(d_b_54, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream54));
    CUDA_CHECK(hipMemcpyAsync(d_a_55, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream55));
    CUDA_CHECK(hipMemcpyAsync(d_b_55, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream55));
    CUDA_CHECK(hipMemcpyAsync(d_a_56, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream56));
    CUDA_CHECK(hipMemcpyAsync(d_b_56, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream56));
    CUDA_CHECK(hipMemcpyAsync(d_a_57, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream57));
    CUDA_CHECK(hipMemcpyAsync(d_b_57, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream57));
    CUDA_CHECK(hipMemcpyAsync(d_a_58, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream58));
    CUDA_CHECK(hipMemcpyAsync(d_b_58, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream58));
    CUDA_CHECK(hipMemcpyAsync(d_a_59, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream59));
    CUDA_CHECK(hipMemcpyAsync(d_b_59, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream59));
    CUDA_CHECK(hipMemcpyAsync(d_a_60, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream60));
    CUDA_CHECK(hipMemcpyAsync(d_b_60, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream60));
    CUDA_CHECK(hipMemcpyAsync(d_a_61, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream61));
    CUDA_CHECK(hipMemcpyAsync(d_b_61, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream61));
    CUDA_CHECK(hipMemcpyAsync(d_a_62, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream62));
    CUDA_CHECK(hipMemcpyAsync(d_b_62, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream62));
    CUDA_CHECK(hipMemcpyAsync(d_a_63, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream63));
    CUDA_CHECK(hipMemcpyAsync(d_b_63, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream63));
    CUDA_CHECK(hipMemcpyAsync(d_a_64, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream64));
    CUDA_CHECK(hipMemcpyAsync(d_b_64, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream64));
    CUDA_CHECK(hipMemcpyAsync(d_a_65, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream65));
    CUDA_CHECK(hipMemcpyAsync(d_b_65, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream65));
    CUDA_CHECK(hipMemcpyAsync(d_a_66, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream66));
    CUDA_CHECK(hipMemcpyAsync(d_b_66, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream66));
    CUDA_CHECK(hipMemcpyAsync(d_a_67, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream67));
    CUDA_CHECK(hipMemcpyAsync(d_b_67, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream67));
    CUDA_CHECK(hipMemcpyAsync(d_a_68, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream68));
    CUDA_CHECK(hipMemcpyAsync(d_b_68, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream68));
    CUDA_CHECK(hipMemcpyAsync(d_a_69, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream69));
    CUDA_CHECK(hipMemcpyAsync(d_b_69, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream69));
    CUDA_CHECK(hipMemcpyAsync(d_a_70, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream70));
    CUDA_CHECK(hipMemcpyAsync(d_b_70, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream70));
    CUDA_CHECK(hipMemcpyAsync(d_a_71, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream71));
    CUDA_CHECK(hipMemcpyAsync(d_b_71, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream71));
    CUDA_CHECK(hipMemcpyAsync(d_a_72, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream72));
    CUDA_CHECK(hipMemcpyAsync(d_b_72, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream72));
    CUDA_CHECK(hipMemcpyAsync(d_a_73, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream73));
    CUDA_CHECK(hipMemcpyAsync(d_b_73, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream73));
    CUDA_CHECK(hipMemcpyAsync(d_a_74, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream74));
    CUDA_CHECK(hipMemcpyAsync(d_b_74, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream74));
    CUDA_CHECK(hipMemcpyAsync(d_a_75, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream75));
    CUDA_CHECK(hipMemcpyAsync(d_b_75, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream75));
    CUDA_CHECK(hipMemcpyAsync(d_a_76, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream76));
    CUDA_CHECK(hipMemcpyAsync(d_b_76, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream76));
    CUDA_CHECK(hipMemcpyAsync(d_a_77, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream77));
    CUDA_CHECK(hipMemcpyAsync(d_b_77, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream77));
    CUDA_CHECK(hipMemcpyAsync(d_a_78, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream78));
    CUDA_CHECK(hipMemcpyAsync(d_b_78, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream78));
    CUDA_CHECK(hipMemcpyAsync(d_a_79, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream79));
    CUDA_CHECK(hipMemcpyAsync(d_b_79, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream79));
    CUDA_CHECK(hipMemcpyAsync(d_a_80, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream80));
    CUDA_CHECK(hipMemcpyAsync(d_b_80, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream80));
    CUDA_CHECK(hipMemcpyAsync(d_a_81, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream81));
    CUDA_CHECK(hipMemcpyAsync(d_b_81, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream81));
    

    // step 3: compute 
    hipblasHgemm(cublasH1, transa, transb, n, m, k, &alpha, d_b_1, ldb, d_a_1, lda, &beta, d_c_1, ldc);hipblasHgemm(cublasH2, transa, transb, n, m, k, &alpha, d_b_2, ldb, d_a_2, lda, &beta, d_c_2, ldc);hipblasHgemm(cublasH3, transa, transb, n, m, k, &alpha, d_b_3, ldb, d_a_3, lda, &beta, d_c_3, ldc);hipblasHgemm(cublasH4, transa, transb, n, m, k, &alpha, d_b_4, ldb, d_a_4, lda, &beta, d_c_4, ldc);hipblasHgemm(cublasH5, transa, transb, n, m, k, &alpha, d_b_5, ldb, d_a_5, lda, &beta, d_c_5, ldc);hipblasHgemm(cublasH6, transa, transb, n, m, k, &alpha, d_b_6, ldb, d_a_6, lda, &beta, d_c_6, ldc);hipblasHgemm(cublasH7, transa, transb, n, m, k, &alpha, d_b_7, ldb, d_a_7, lda, &beta, d_c_7, ldc);hipblasHgemm(cublasH8, transa, transb, n, m, k, &alpha, d_b_8, ldb, d_a_8, lda, &beta, d_c_8, ldc);hipblasHgemm(cublasH9, transa, transb, n, m, k, &alpha, d_b_9, ldb, d_a_9, lda, &beta, d_c_9, ldc);hipblasHgemm(cublasH10, transa, transb, n, m, k, &alpha, d_b_10, ldb, d_a_10, lda, &beta, d_c_10, ldc);hipblasHgemm(cublasH11, transa, transb, n, m, k, &alpha, d_b_11, ldb, d_a_11, lda, &beta, d_c_11, ldc);hipblasHgemm(cublasH12, transa, transb, n, m, k, &alpha, d_b_12, ldb, d_a_12, lda, &beta, d_c_12, ldc);hipblasHgemm(cublasH13, transa, transb, n, m, k, &alpha, d_b_13, ldb, d_a_13, lda, &beta, d_c_13, ldc);hipblasHgemm(cublasH14, transa, transb, n, m, k, &alpha, d_b_14, ldb, d_a_14, lda, &beta, d_c_14, ldc);hipblasHgemm(cublasH15, transa, transb, n, m, k, &alpha, d_b_15, ldb, d_a_15, lda, &beta, d_c_15, ldc);hipblasHgemm(cublasH16, transa, transb, n, m, k, &alpha, d_b_16, ldb, d_a_16, lda, &beta, d_c_16, ldc);hipblasHgemm(cublasH17, transa, transb, n, m, k, &alpha, d_b_17, ldb, d_a_17, lda, &beta, d_c_17, ldc);hipblasHgemm(cublasH18, transa, transb, n, m, k, &alpha, d_b_18, ldb, d_a_18, lda, &beta, d_c_18, ldc);hipblasHgemm(cublasH19, transa, transb, n, m, k, &alpha, d_b_19, ldb, d_a_19, lda, &beta, d_c_19, ldc);hipblasHgemm(cublasH20, transa, transb, n, m, k, &alpha, d_b_20, ldb, d_a_20, lda, &beta, d_c_20, ldc);hipblasHgemm(cublasH21, transa, transb, n, m, k, &alpha, d_b_21, ldb, d_a_21, lda, &beta, d_c_21, ldc);hipblasHgemm(cublasH22, transa, transb, n, m, k, &alpha, d_b_22, ldb, d_a_22, lda, &beta, d_c_22, ldc);hipblasHgemm(cublasH23, transa, transb, n, m, k, &alpha, d_b_23, ldb, d_a_23, lda, &beta, d_c_23, ldc);hipblasHgemm(cublasH24, transa, transb, n, m, k, &alpha, d_b_24, ldb, d_a_24, lda, &beta, d_c_24, ldc);hipblasHgemm(cublasH25, transa, transb, n, m, k, &alpha, d_b_25, ldb, d_a_25, lda, &beta, d_c_25, ldc);hipblasHgemm(cublasH26, transa, transb, n, m, k, &alpha, d_b_26, ldb, d_a_26, lda, &beta, d_c_26, ldc);hipblasHgemm(cublasH27, transa, transb, n, m, k, &alpha, d_b_27, ldb, d_a_27, lda, &beta, d_c_27, ldc);hipblasHgemm(cublasH28, transa, transb, n, m, k, &alpha, d_b_28, ldb, d_a_28, lda, &beta, d_c_28, ldc);hipblasHgemm(cublasH29, transa, transb, n, m, k, &alpha, d_b_29, ldb, d_a_29, lda, &beta, d_c_29, ldc);hipblasHgemm(cublasH30, transa, transb, n, m, k, &alpha, d_b_30, ldb, d_a_30, lda, &beta, d_c_30, ldc);hipblasHgemm(cublasH31, transa, transb, n, m, k, &alpha, d_b_31, ldb, d_a_31, lda, &beta, d_c_31, ldc);hipblasHgemm(cublasH32, transa, transb, n, m, k, &alpha, d_b_32, ldb, d_a_32, lda, &beta, d_c_32, ldc);hipblasHgemm(cublasH33, transa, transb, n, m, k, &alpha, d_b_33, ldb, d_a_33, lda, &beta, d_c_33, ldc);hipblasHgemm(cublasH34, transa, transb, n, m, k, &alpha, d_b_34, ldb, d_a_34, lda, &beta, d_c_34, ldc);hipblasHgemm(cublasH35, transa, transb, n, m, k, &alpha, d_b_35, ldb, d_a_35, lda, &beta, d_c_35, ldc);hipblasHgemm(cublasH36, transa, transb, n, m, k, &alpha, d_b_36, ldb, d_a_36, lda, &beta, d_c_36, ldc);hipblasHgemm(cublasH37, transa, transb, n, m, k, &alpha, d_b_37, ldb, d_a_37, lda, &beta, d_c_37, ldc);hipblasHgemm(cublasH38, transa, transb, n, m, k, &alpha, d_b_38, ldb, d_a_38, lda, &beta, d_c_38, ldc);hipblasHgemm(cublasH39, transa, transb, n, m, k, &alpha, d_b_39, ldb, d_a_39, lda, &beta, d_c_39, ldc);hipblasHgemm(cublasH40, transa, transb, n, m, k, &alpha, d_b_40, ldb, d_a_40, lda, &beta, d_c_40, ldc);hipblasHgemm(cublasH41, transa, transb, n, m, k, &alpha, d_b_41, ldb, d_a_41, lda, &beta, d_c_41, ldc);hipblasHgemm(cublasH42, transa, transb, n, m, k, &alpha, d_b_42, ldb, d_a_42, lda, &beta, d_c_42, ldc);hipblasHgemm(cublasH43, transa, transb, n, m, k, &alpha, d_b_43, ldb, d_a_43, lda, &beta, d_c_43, ldc);hipblasHgemm(cublasH44, transa, transb, n, m, k, &alpha, d_b_44, ldb, d_a_44, lda, &beta, d_c_44, ldc);hipblasHgemm(cublasH45, transa, transb, n, m, k, &alpha, d_b_45, ldb, d_a_45, lda, &beta, d_c_45, ldc);hipblasHgemm(cublasH46, transa, transb, n, m, k, &alpha, d_b_46, ldb, d_a_46, lda, &beta, d_c_46, ldc);hipblasHgemm(cublasH47, transa, transb, n, m, k, &alpha, d_b_47, ldb, d_a_47, lda, &beta, d_c_47, ldc);hipblasHgemm(cublasH48, transa, transb, n, m, k, &alpha, d_b_48, ldb, d_a_48, lda, &beta, d_c_48, ldc);hipblasHgemm(cublasH49, transa, transb, n, m, k, &alpha, d_b_49, ldb, d_a_49, lda, &beta, d_c_49, ldc);hipblasHgemm(cublasH50, transa, transb, n, m, k, &alpha, d_b_50, ldb, d_a_50, lda, &beta, d_c_50, ldc);hipblasHgemm(cublasH51, transa, transb, n, m, k, &alpha, d_b_51, ldb, d_a_51, lda, &beta, d_c_51, ldc);hipblasHgemm(cublasH52, transa, transb, n, m, k, &alpha, d_b_52, ldb, d_a_52, lda, &beta, d_c_52, ldc);hipblasHgemm(cublasH53, transa, transb, n, m, k, &alpha, d_b_53, ldb, d_a_53, lda, &beta, d_c_53, ldc);hipblasHgemm(cublasH54, transa, transb, n, m, k, &alpha, d_b_54, ldb, d_a_54, lda, &beta, d_c_54, ldc);hipblasHgemm(cublasH55, transa, transb, n, m, k, &alpha, d_b_55, ldb, d_a_55, lda, &beta, d_c_55, ldc);hipblasHgemm(cublasH56, transa, transb, n, m, k, &alpha, d_b_56, ldb, d_a_56, lda, &beta, d_c_56, ldc);hipblasHgemm(cublasH57, transa, transb, n, m, k, &alpha, d_b_57, ldb, d_a_57, lda, &beta, d_c_57, ldc);hipblasHgemm(cublasH58, transa, transb, n, m, k, &alpha, d_b_58, ldb, d_a_58, lda, &beta, d_c_58, ldc);hipblasHgemm(cublasH59, transa, transb, n, m, k, &alpha, d_b_59, ldb, d_a_59, lda, &beta, d_c_59, ldc);hipblasHgemm(cublasH60, transa, transb, n, m, k, &alpha, d_b_60, ldb, d_a_60, lda, &beta, d_c_60, ldc);hipblasHgemm(cublasH61, transa, transb, n, m, k, &alpha, d_b_61, ldb, d_a_61, lda, &beta, d_c_61, ldc);hipblasHgemm(cublasH62, transa, transb, n, m, k, &alpha, d_b_62, ldb, d_a_62, lda, &beta, d_c_62, ldc);hipblasHgemm(cublasH63, transa, transb, n, m, k, &alpha, d_b_63, ldb, d_a_63, lda, &beta, d_c_63, ldc);hipblasHgemm(cublasH64, transa, transb, n, m, k, &alpha, d_b_64, ldb, d_a_64, lda, &beta, d_c_64, ldc);hipblasHgemm(cublasH65, transa, transb, n, m, k, &alpha, d_b_65, ldb, d_a_65, lda, &beta, d_c_65, ldc);hipblasHgemm(cublasH66, transa, transb, n, m, k, &alpha, d_b_66, ldb, d_a_66, lda, &beta, d_c_66, ldc);hipblasHgemm(cublasH67, transa, transb, n, m, k, &alpha, d_b_67, ldb, d_a_67, lda, &beta, d_c_67, ldc);hipblasHgemm(cublasH68, transa, transb, n, m, k, &alpha, d_b_68, ldb, d_a_68, lda, &beta, d_c_68, ldc);hipblasHgemm(cublasH69, transa, transb, n, m, k, &alpha, d_b_69, ldb, d_a_69, lda, &beta, d_c_69, ldc);hipblasHgemm(cublasH70, transa, transb, n, m, k, &alpha, d_b_70, ldb, d_a_70, lda, &beta, d_c_70, ldc);hipblasHgemm(cublasH71, transa, transb, n, m, k, &alpha, d_b_71, ldb, d_a_71, lda, &beta, d_c_71, ldc);hipblasHgemm(cublasH72, transa, transb, n, m, k, &alpha, d_b_72, ldb, d_a_72, lda, &beta, d_c_72, ldc);hipblasHgemm(cublasH73, transa, transb, n, m, k, &alpha, d_b_73, ldb, d_a_73, lda, &beta, d_c_73, ldc);hipblasHgemm(cublasH74, transa, transb, n, m, k, &alpha, d_b_74, ldb, d_a_74, lda, &beta, d_c_74, ldc);hipblasHgemm(cublasH75, transa, transb, n, m, k, &alpha, d_b_75, ldb, d_a_75, lda, &beta, d_c_75, ldc);hipblasHgemm(cublasH76, transa, transb, n, m, k, &alpha, d_b_76, ldb, d_a_76, lda, &beta, d_c_76, ldc);hipblasHgemm(cublasH77, transa, transb, n, m, k, &alpha, d_b_77, ldb, d_a_77, lda, &beta, d_c_77, ldc);hipblasHgemm(cublasH78, transa, transb, n, m, k, &alpha, d_b_78, ldb, d_a_78, lda, &beta, d_c_78, ldc);hipblasHgemm(cublasH79, transa, transb, n, m, k, &alpha, d_b_79, ldb, d_a_79, lda, &beta, d_c_79, ldc);hipblasHgemm(cublasH80, transa, transb, n, m, k, &alpha, d_b_80, ldb, d_a_80, lda, &beta, d_c_80, ldc);hipblasHgemm(cublasH81, transa, transb, n, m, k, &alpha, d_b_81, ldb, d_a_81, lda, &beta, d_c_81, ldc);

    // cublasHgemm111<<<1,1,1,stream1>>>(cublasH1, transa, transb, n, m, k, &alpha, d_b_1, ldb, d_a_1, lda, &beta, d_c_1, ldc);
    // cublasHgemm111<<<2,1,1,stream2>>>(cublasH2, transa, transb, n, m, k, &alpha, d_b_2, ldb, d_a_2, lda, &beta, d_c_2, ldc);
    // cublasHgemm111<<<3,1,1,stream3>>>(cublasH3, transa, transb, n, m, k, &alpha, d_b_3, ldb, d_a_3, lda, &beta, d_c_3, ldc);
    // cublasHgemm111<<<4,1,1,stream4>>>(cublasH4, transa, transb, n, m, k, &alpha, d_b_4, ldb, d_a_4, lda, &beta, d_c_4, ldc);
    // cublasHgemm111<<<5,1,1,stream5>>>(cublasH5, transa, transb, n, m, k, &alpha, d_b_5, ldb, d_a_5, lda, &beta, d_c_5, ldc);
    // cublasHgemm111<<<6,1,1,stream6>>>(cublasH6, transa, transb, n, m, k, &alpha, d_b_6, ldb, d_a_6, lda, &beta, d_c_6, ldc);
    // cublasHgemm111<<<7,1,1,stream7>>>(cublasH7, transa, transb, n, m, k, &alpha, d_b_7, ldb, d_a_7, lda, &beta, d_c_7, ldc);
    // cublasHgemm111<<<8,1,1,stream8>>>(cublasH8, transa, transb, n, m, k, &alpha, d_b_8, ldb, d_a_8, lda, &beta, d_c_8, ldc);
    // cublasHgemm111<<<9,1,1,stream9>>>(cublasH9, transa, transb, n, m, k, &alpha, d_b_9, ldb, d_a_9, lda, &beta, d_c_9, ldc);
    // cublasHgemm111<<<10,1,1,stream10>>>(cublasH10, transa, transb, n, m, k, &alpha, d_b_10, ldb, d_a_10, lda, &beta, d_c_10, ldc);
    // cublasHgemm111<<<11,1,1,stream11>>>(cublasH11, transa, transb, n, m, k, &alpha, d_b_11, ldb, d_a_11, lda, &beta, d_c_11, ldc);
    // cublasHgemm111<<<12,1,1,stream12>>>(cublasH12, transa, transb, n, m, k, &alpha, d_b_12, ldb, d_a_12, lda, &beta, d_c_12, ldc);
    // cublasHgemm111<<<13,1,1,stream13>>>(cublasH13, transa, transb, n, m, k, &alpha, d_b_13, ldb, d_a_13, lda, &beta, d_c_13, ldc);
    // cublasHgemm111<<<14,1,1,stream14>>>(cublasH14, transa, transb, n, m, k, &alpha, d_b_14, ldb, d_a_14, lda, &beta, d_c_14, ldc);
    // cublasHgemm111<<<15,1,1,stream15>>>(cublasH15, transa, transb, n, m, k, &alpha, d_b_15, ldb, d_a_15, lda, &beta, d_c_15, ldc);
    // cublasHgemm111<<<16,1,1,stream16>>>(cublasH16, transa, transb, n, m, k, &alpha, d_b_16, ldb, d_a_16, lda, &beta, d_c_16, ldc);
    // cublasHgemm111<<<17,1,1,stream17>>>(cublasH17, transa, transb, n, m, k, &alpha, d_b_17, ldb, d_a_17, lda, &beta, d_c_17, ldc);
    // cublasHgemm111<<<18,1,1,stream18>>>(cublasH18, transa, transb, n, m, k, &alpha, d_b_18, ldb, d_a_18, lda, &beta, d_c_18, ldc);
    // cublasHgemm111<<<19,1,1,stream19>>>(cublasH19, transa, transb, n, m, k, &alpha, d_b_19, ldb, d_a_19, lda, &beta, d_c_19, ldc);
    // cublasHgemm111<<<20,1,1,stream20>>>(cublasH20, transa, transb, n, m, k, &alpha, d_b_20, ldb, d_a_20, lda, &beta, d_c_20, ldc);
    // cublasHgemm111<<<21,1,1,stream21>>>(cublasH21, transa, transb, n, m, k, &alpha, d_b_21, ldb, d_a_21, lda, &beta, d_c_21, ldc);
    // cublasHgemm111<<<22,1,1,stream22>>>(cublasH22, transa, transb, n, m, k, &alpha, d_b_22, ldb, d_a_22, lda, &beta, d_c_22, ldc);
    // cublasHgemm111<<<23,1,1,stream23>>>(cublasH23, transa, transb, n, m, k, &alpha, d_b_23, ldb, d_a_23, lda, &beta, d_c_23, ldc);
    // cublasHgemm111<<<24,1,1,stream24>>>(cublasH24, transa, transb, n, m, k, &alpha, d_b_24, ldb, d_a_24, lda, &beta, d_c_24, ldc);
    // cublasHgemm111<<<25,1,1,stream25>>>(cublasH25, transa, transb, n, m, k, &alpha, d_b_25, ldb, d_a_25, lda, &beta, d_c_25, ldc);
    // cublasHgemm111<<<26,1,1,stream26>>>(cublasH26, transa, transb, n, m, k, &alpha, d_b_26, ldb, d_a_26, lda, &beta, d_c_26, ldc);
    // cublasHgemm111<<<27,1,1,stream27>>>(cublasH27, transa, transb, n, m, k, &alpha, d_b_27, ldb, d_a_27, lda, &beta, d_c_27, ldc);
    // cublasHgemm111<<<28,1,1,stream28>>>(cublasH28, transa, transb, n, m, k, &alpha, d_b_28, ldb, d_a_28, lda, &beta, d_c_28, ldc);
    // cublasHgemm111<<<29,1,1,stream29>>>(cublasH29, transa, transb, n, m, k, &alpha, d_b_29, ldb, d_a_29, lda, &beta, d_c_29, ldc);
    // cublasHgemm111<<<30,1,1,stream30>>>(cublasH30, transa, transb, n, m, k, &alpha, d_b_30, ldb, d_a_30, lda, &beta, d_c_30, ldc);
    // cublasHgemm111<<<31,1,1,stream31>>>(cublasH31, transa, transb, n, m, k, &alpha, d_b_31, ldb, d_a_31, lda, &beta, d_c_31, ldc);
    // cublasHgemm111<<<32,1,1,stream32>>>(cublasH32, transa, transb, n, m, k, &alpha, d_b_32, ldb, d_a_32, lda, &beta, d_c_32, ldc);
    // cublasHgemm111<<<33,1,1,stream33>>>(cublasH33, transa, transb, n, m, k, &alpha, d_b_33, ldb, d_a_33, lda, &beta, d_c_33, ldc);
    // cublasHgemm111<<<34,1,1,stream34>>>(cublasH34, transa, transb, n, m, k, &alpha, d_b_34, ldb, d_a_34, lda, &beta, d_c_34, ldc);
    // cublasHgemm111<<<35,1,1,stream35>>>(cublasH35, transa, transb, n, m, k, &alpha, d_b_35, ldb, d_a_35, lda, &beta, d_c_35, ldc);
    // cublasHgemm111<<<36,1,1,stream36>>>(cublasH36, transa, transb, n, m, k, &alpha, d_b_36, ldb, d_a_36, lda, &beta, d_c_36, ldc);
    // cublasHgemm111<<<37,1,1,stream37>>>(cublasH37, transa, transb, n, m, k, &alpha, d_b_37, ldb, d_a_37, lda, &beta, d_c_37, ldc);
    // cublasHgemm111<<<38,1,1,stream38>>>(cublasH38, transa, transb, n, m, k, &alpha, d_b_38, ldb, d_a_38, lda, &beta, d_c_38, ldc);
    // cublasHgemm111<<<39,1,1,stream39>>>(cublasH39, transa, transb, n, m, k, &alpha, d_b_39, ldb, d_a_39, lda, &beta, d_c_39, ldc);
    // cublasHgemm111<<<40,1,1,stream40>>>(cublasH40, transa, transb, n, m, k, &alpha, d_b_40, ldb, d_a_40, lda, &beta, d_c_40, ldc);
    // cublasHgemm111<<<41,1,1,stream41>>>(cublasH41, transa, transb, n, m, k, &alpha, d_b_41, ldb, d_a_41, lda, &beta, d_c_41, ldc);
    // cublasHgemm111<<<42,1,1,stream42>>>(cublasH42, transa, transb, n, m, k, &alpha, d_b_42, ldb, d_a_42, lda, &beta, d_c_42, ldc);
    // cublasHgemm111<<<43,1,1,stream43>>>(cublasH43, transa, transb, n, m, k, &alpha, d_b_43, ldb, d_a_43, lda, &beta, d_c_43, ldc);
    // cublasHgemm111<<<44,1,1,stream44>>>(cublasH44, transa, transb, n, m, k, &alpha, d_b_44, ldb, d_a_44, lda, &beta, d_c_44, ldc);
    // cublasHgemm111<<<43,1,1,stream45>>>(cublasH45, transa, transb, n, m, k, &alpha, d_b_45, ldb, d_a_45, lda, &beta, d_c_45, ldc);
    // cublasHgemm111<<<42,1,1,stream46>>>(cublasH46, transa, transb, n, m, k, &alpha, d_b_46, ldb, d_a_46, lda, &beta, d_c_46, ldc);
    // cublasHgemm111<<<41,1,1,stream47>>>(cublasH47, transa, transb, n, m, k, &alpha, d_b_47, ldb, d_a_47, lda, &beta, d_c_47, ldc);
    // cublasHgemm111<<<40,1,1,stream48>>>(cublasH48, transa, transb, n, m, k, &alpha, d_b_48, ldb, d_a_48, lda, &beta, d_c_48, ldc);
    // cublasHgemm111<<<39,1,1,stream49>>>(cublasH49, transa, transb, n, m, k, &alpha, d_b_49, ldb, d_a_49, lda, &beta, d_c_49, ldc);
    // cublasHgemm111<<<38,1,1,stream50>>>(cublasH50, transa, transb, n, m, k, &alpha, d_b_50, ldb, d_a_50, lda, &beta, d_c_50, ldc);
    // cublasHgemm111<<<37,1,1,stream51>>>(cublasH51, transa, transb, n, m, k, &alpha, d_b_51, ldb, d_a_51, lda, &beta, d_c_51, ldc);
    // cublasHgemm111<<<36,1,1,stream52>>>(cublasH52, transa, transb, n, m, k, &alpha, d_b_52, ldb, d_a_52, lda, &beta, d_c_52, ldc);
    // cublasHgemm111<<<35,1,1,stream53>>>(cublasH53, transa, transb, n, m, k, &alpha, d_b_53, ldb, d_a_53, lda, &beta, d_c_53, ldc);
    // cublasHgemm111<<<34,1,1,stream54>>>(cublasH54, transa, transb, n, m, k, &alpha, d_b_54, ldb, d_a_54, lda, &beta, d_c_54, ldc);
    // cublasHgemm111<<<33,1,1,stream55>>>(cublasH55, transa, transb, n, m, k, &alpha, d_b_55, ldb, d_a_55, lda, &beta, d_c_55, ldc);
    // cublasHgemm111<<<32,1,1,stream56>>>(cublasH56, transa, transb, n, m, k, &alpha, d_b_56, ldb, d_a_56, lda, &beta, d_c_56, ldc);
    // cublasHgemm111<<<31,1,1,stream57>>>(cublasH57, transa, transb, n, m, k, &alpha, d_b_57, ldb, d_a_57, lda, &beta, d_c_57, ldc);
    // cublasHgemm111<<<30,1,1,stream58>>>(cublasH58, transa, transb, n, m, k, &alpha, d_b_58, ldb, d_a_58, lda, &beta, d_c_58, ldc);
    // cublasHgemm111<<<29,1,1,stream59>>>(cublasH59, transa, transb, n, m, k, &alpha, d_b_59, ldb, d_a_59, lda, &beta, d_c_59, ldc);
    // cublasHgemm111<<<28,1,1,stream60>>>(cublasH60, transa, transb, n, m, k, &alpha, d_b_60, ldb, d_a_60, lda, &beta, d_c_60, ldc);
    // cublasHgemm111<<<27,1,1,stream61>>>(cublasH61, transa, transb, n, m, k, &alpha, d_b_61, ldb, d_a_61, lda, &beta, d_c_61, ldc);
    // cublasHgemm111<<<26,1,1,stream62>>>(cublasH62, transa, transb, n, m, k, &alpha, d_b_62, ldb, d_a_62, lda, &beta, d_c_62, ldc);
    // cublasHgemm111<<<25,1,1,stream63>>>(cublasH63, transa, transb, n, m, k, &alpha, d_b_63, ldb, d_a_63, lda, &beta, d_c_63, ldc);
    // cublasHgemm111<<<24,1,1,stream64>>>(cublasH64, transa, transb, n, m, k, &alpha, d_b_64, ldb, d_a_64, lda, &beta, d_c_64, ldc);
    // cublasHgemm111<<<23,1,1,stream65>>>(cublasH65, transa, transb, n, m, k, &alpha, d_b_65, ldb, d_a_65, lda, &beta, d_c_65, ldc);
    // cublasHgemm111<<<22,1,1,stream66>>>(cublasH66, transa, transb, n, m, k, &alpha, d_b_66, ldb, d_a_66, lda, &beta, d_c_66, ldc);
    // cublasHgemm111<<<21,1,1,stream67>>>(cublasH67, transa, transb, n, m, k, &alpha, d_b_67, ldb, d_a_67, lda, &beta, d_c_67, ldc);
    // cublasHgemm111<<<20,1,1,stream68>>>(cublasH68, transa, transb, n, m, k, &alpha, d_b_68, ldb, d_a_68, lda, &beta, d_c_68, ldc);
    // cublasHgemm111<<<19,1,1,stream69>>>(cublasH69, transa, transb, n, m, k, &alpha, d_b_69, ldb, d_a_69, lda, &beta, d_c_69, ldc);
    // cublasHgemm111<<<18,1,1,stream70>>>(cublasH70, transa, transb, n, m, k, &alpha, d_b_70, ldb, d_a_70, lda, &beta, d_c_70, ldc);
    // cublasHgemm111<<<17,1,1,stream71>>>(cublasH71, transa, transb, n, m, k, &alpha, d_b_71, ldb, d_a_71, lda, &beta, d_c_71, ldc);
    // cublasHgemm111<<<16,1,1,stream72>>>(cublasH72, transa, transb, n, m, k, &alpha, d_b_72, ldb, d_a_72, lda, &beta, d_c_72, ldc);
    // cublasHgemm111<<<15,1,1,stream73>>>(cublasH73, transa, transb, n, m, k, &alpha, d_b_73, ldb, d_a_73, lda, &beta, d_c_73, ldc);
    // cublasHgemm111<<<14,1,1,stream74>>>(cublasH74, transa, transb, n, m, k, &alpha, d_b_74, ldb, d_a_74, lda, &beta, d_c_74, ldc);
    // cublasHgemm111<<<13,1,1,stream75>>>(cublasH75, transa, transb, n, m, k, &alpha, d_b_75, ldb, d_a_75, lda, &beta, d_c_75, ldc);
    // cublasHgemm111<<<12,1,1,stream76>>>(cublasH76, transa, transb, n, m, k, &alpha, d_b_76, ldb, d_a_76, lda, &beta, d_c_76, ldc);
    // cublasHgemm111<<<11,1,1,stream77>>>(cublasH77, transa, transb, n, m, k, &alpha, d_b_77, ldb, d_a_77, lda, &beta, d_c_77, ldc);
    // cublasHgemm111<<<10,1,1,stream78>>>(cublasH78, transa, transb, n, m, k, &alpha, d_b_78, ldb, d_a_78, lda, &beta, d_c_78, ldc);
    // cublasHgemm111<<<9,1,1,stream79>>>(cublasH79, transa, transb, n, m, k, &alpha, d_b_79, ldb, d_a_79, lda, &beta, d_c_79, ldc);
    // cublasHgemm111<<<8,1,1,stream80>>>(cublasH80, transa, transb, n, m, k, &alpha, d_b_80, ldb, d_a_80, lda, &beta, d_c_80, ldc);
    // cublasHgemm111<<<7,1,1,stream81>>>(cublasH81, transa, transb, n, m, k, &alpha, d_b_81, ldb, d_a_81, lda, &beta, d_c_81, ldc);
    


    // step 4: copy data to host 
    // CUDA_CHECK(cudaMemcpyAsync(C.data(), d_c_1, sizeof(data_type) * C.size(), cudaMemcpyDeviceToHost,
    //                            stream));

    // cudaStreamSynchronize 71 streams
    CUDA_CHECK(hipStreamSynchronize(stream1));CUDA_CHECK(hipStreamSynchronize(stream2));CUDA_CHECK(hipStreamSynchronize(stream3));CUDA_CHECK(hipStreamSynchronize(stream4));CUDA_CHECK(hipStreamSynchronize(stream5));CUDA_CHECK(hipStreamSynchronize(stream6));CUDA_CHECK(hipStreamSynchronize(stream7));CUDA_CHECK(hipStreamSynchronize(stream8));CUDA_CHECK(hipStreamSynchronize(stream9));CUDA_CHECK(hipStreamSynchronize(stream10));CUDA_CHECK(hipStreamSynchronize(stream11));CUDA_CHECK(hipStreamSynchronize(stream12));CUDA_CHECK(hipStreamSynchronize(stream13));CUDA_CHECK(hipStreamSynchronize(stream14));CUDA_CHECK(hipStreamSynchronize(stream15));CUDA_CHECK(hipStreamSynchronize(stream16));CUDA_CHECK(hipStreamSynchronize(stream17));CUDA_CHECK(hipStreamSynchronize(stream18));CUDA_CHECK(hipStreamSynchronize(stream19));CUDA_CHECK(hipStreamSynchronize(stream20));CUDA_CHECK(hipStreamSynchronize(stream21));CUDA_CHECK(hipStreamSynchronize(stream22));CUDA_CHECK(hipStreamSynchronize(stream23));CUDA_CHECK(hipStreamSynchronize(stream24));CUDA_CHECK(hipStreamSynchronize(stream25));CUDA_CHECK(hipStreamSynchronize(stream26));CUDA_CHECK(hipStreamSynchronize(stream27));CUDA_CHECK(hipStreamSynchronize(stream28));CUDA_CHECK(hipStreamSynchronize(stream29));CUDA_CHECK(hipStreamSynchronize(stream30));CUDA_CHECK(hipStreamSynchronize(stream31));CUDA_CHECK(hipStreamSynchronize(stream32));CUDA_CHECK(hipStreamSynchronize(stream33));CUDA_CHECK(hipStreamSynchronize(stream34));CUDA_CHECK(hipStreamSynchronize(stream35));CUDA_CHECK(hipStreamSynchronize(stream36));CUDA_CHECK(hipStreamSynchronize(stream37));CUDA_CHECK(hipStreamSynchronize(stream38));CUDA_CHECK(hipStreamSynchronize(stream39));CUDA_CHECK(hipStreamSynchronize(stream40));CUDA_CHECK(hipStreamSynchronize(stream41));CUDA_CHECK(hipStreamSynchronize(stream42));CUDA_CHECK(hipStreamSynchronize(stream43));CUDA_CHECK(hipStreamSynchronize(stream44));CUDA_CHECK(hipStreamSynchronize(stream45));CUDA_CHECK(hipStreamSynchronize(stream46));CUDA_CHECK(hipStreamSynchronize(stream47));CUDA_CHECK(hipStreamSynchronize(stream48));CUDA_CHECK(hipStreamSynchronize(stream49));CUDA_CHECK(hipStreamSynchronize(stream50));CUDA_CHECK(hipStreamSynchronize(stream51));CUDA_CHECK(hipStreamSynchronize(stream52));CUDA_CHECK(hipStreamSynchronize(stream53));CUDA_CHECK(hipStreamSynchronize(stream54));CUDA_CHECK(hipStreamSynchronize(stream55));CUDA_CHECK(hipStreamSynchronize(stream56));CUDA_CHECK(hipStreamSynchronize(stream57));CUDA_CHECK(hipStreamSynchronize(stream58));CUDA_CHECK(hipStreamSynchronize(stream59));CUDA_CHECK(hipStreamSynchronize(stream60));CUDA_CHECK(hipStreamSynchronize(stream61));CUDA_CHECK(hipStreamSynchronize(stream62));CUDA_CHECK(hipStreamSynchronize(stream63));CUDA_CHECK(hipStreamSynchronize(stream64));CUDA_CHECK(hipStreamSynchronize(stream65));CUDA_CHECK(hipStreamSynchronize(stream66));CUDA_CHECK(hipStreamSynchronize(stream67));CUDA_CHECK(hipStreamSynchronize(stream68));CUDA_CHECK(hipStreamSynchronize(stream69));CUDA_CHECK(hipStreamSynchronize(stream70));CUDA_CHECK(hipStreamSynchronize(stream71));CUDA_CHECK(hipStreamSynchronize(stream72));CUDA_CHECK(hipStreamSynchronize(stream73));CUDA_CHECK(hipStreamSynchronize(stream74));CUDA_CHECK(hipStreamSynchronize(stream75));CUDA_CHECK(hipStreamSynchronize(stream76));CUDA_CHECK(hipStreamSynchronize(stream77));CUDA_CHECK(hipStreamSynchronize(stream78));CUDA_CHECK(hipStreamSynchronize(stream79));CUDA_CHECK(hipStreamSynchronize(stream80));CUDA_CHECK(hipStreamSynchronize(stream81));

    //
    //   C = | 19.0 | 22.0 |
    //       | 43.0 | 50.0 |
    //
    
    // free resources 
    CUDA_CHECK(hipFree(d_a_1));
    CUDA_CHECK(hipFree(d_b_1));
    CUDA_CHECK(hipFree(d_c_1));
    CUDA_CHECK(hipFree(d_a_2));
    CUDA_CHECK(hipFree(d_b_2));
    CUDA_CHECK(hipFree(d_c_2));
    CUDA_CHECK(hipFree(d_a_3));
    CUDA_CHECK(hipFree(d_b_3));
    CUDA_CHECK(hipFree(d_c_3));
    CUDA_CHECK(hipFree(d_a_4));
    CUDA_CHECK(hipFree(d_b_4));
    CUDA_CHECK(hipFree(d_c_4));
    CUDA_CHECK(hipFree(d_a_5));
    CUDA_CHECK(hipFree(d_b_5));
    CUDA_CHECK(hipFree(d_c_5));
    CUDA_CHECK(hipFree(d_a_6));
    CUDA_CHECK(hipFree(d_b_6));
    CUDA_CHECK(hipFree(d_c_6));
    CUDA_CHECK(hipFree(d_a_7));
    CUDA_CHECK(hipFree(d_b_7));
    CUDA_CHECK(hipFree(d_c_7));
    CUDA_CHECK(hipFree(d_a_8));
    CUDA_CHECK(hipFree(d_b_8));
    CUDA_CHECK(hipFree(d_c_8));
    CUDA_CHECK(hipFree(d_a_9));
    CUDA_CHECK(hipFree(d_b_9));
    CUDA_CHECK(hipFree(d_c_9));
    CUDA_CHECK(hipFree(d_a_10));
    CUDA_CHECK(hipFree(d_b_10));
    CUDA_CHECK(hipFree(d_c_10));
    CUDA_CHECK(hipFree(d_a_11));
    CUDA_CHECK(hipFree(d_b_11));
    CUDA_CHECK(hipFree(d_c_11));
    CUDA_CHECK(hipFree(d_a_12));
    CUDA_CHECK(hipFree(d_b_12));
    CUDA_CHECK(hipFree(d_c_12));
    CUDA_CHECK(hipFree(d_a_13));
    CUDA_CHECK(hipFree(d_b_13));
    CUDA_CHECK(hipFree(d_c_13));
    CUDA_CHECK(hipFree(d_a_14));
    CUDA_CHECK(hipFree(d_b_14));
    CUDA_CHECK(hipFree(d_c_14));
    CUDA_CHECK(hipFree(d_a_15));
    CUDA_CHECK(hipFree(d_b_15));
    CUDA_CHECK(hipFree(d_c_15));
    CUDA_CHECK(hipFree(d_a_16));
    CUDA_CHECK(hipFree(d_b_16));
    CUDA_CHECK(hipFree(d_c_16));
    CUDA_CHECK(hipFree(d_a_17));
    CUDA_CHECK(hipFree(d_b_17));
    CUDA_CHECK(hipFree(d_c_17));
    CUDA_CHECK(hipFree(d_a_18));
    CUDA_CHECK(hipFree(d_b_18));
    CUDA_CHECK(hipFree(d_c_18));
    CUDA_CHECK(hipFree(d_a_19));
    CUDA_CHECK(hipFree(d_b_19));
    CUDA_CHECK(hipFree(d_c_19));
    CUDA_CHECK(hipFree(d_a_20));
    CUDA_CHECK(hipFree(d_b_20));
    CUDA_CHECK(hipFree(d_c_20));
    CUDA_CHECK(hipFree(d_a_21));
    CUDA_CHECK(hipFree(d_b_21));
    CUDA_CHECK(hipFree(d_c_21));
    CUDA_CHECK(hipFree(d_a_22));
    CUDA_CHECK(hipFree(d_b_22));
    CUDA_CHECK(hipFree(d_c_22));
    CUDA_CHECK(hipFree(d_a_23));
    CUDA_CHECK(hipFree(d_b_23));
    CUDA_CHECK(hipFree(d_c_23));
    CUDA_CHECK(hipFree(d_a_24));
    CUDA_CHECK(hipFree(d_b_24));
    CUDA_CHECK(hipFree(d_c_24));
    CUDA_CHECK(hipFree(d_a_25));
    CUDA_CHECK(hipFree(d_b_25));
    CUDA_CHECK(hipFree(d_c_25));
    CUDA_CHECK(hipFree(d_a_26));
    CUDA_CHECK(hipFree(d_b_26));
    CUDA_CHECK(hipFree(d_c_26));
    CUDA_CHECK(hipFree(d_a_27));
    CUDA_CHECK(hipFree(d_b_27));
    CUDA_CHECK(hipFree(d_c_27));
    CUDA_CHECK(hipFree(d_a_28));
    CUDA_CHECK(hipFree(d_b_28));
    CUDA_CHECK(hipFree(d_c_28));
    CUDA_CHECK(hipFree(d_a_29));
    CUDA_CHECK(hipFree(d_b_29));
    CUDA_CHECK(hipFree(d_c_29));
    CUDA_CHECK(hipFree(d_a_30));
    CUDA_CHECK(hipFree(d_b_30));
    CUDA_CHECK(hipFree(d_c_30));
    CUDA_CHECK(hipFree(d_a_31));
    CUDA_CHECK(hipFree(d_b_31));
    CUDA_CHECK(hipFree(d_c_31));
    CUDA_CHECK(hipFree(d_a_32));
    CUDA_CHECK(hipFree(d_b_32));
    CUDA_CHECK(hipFree(d_c_32));
    CUDA_CHECK(hipFree(d_a_33));
    CUDA_CHECK(hipFree(d_b_33));
    CUDA_CHECK(hipFree(d_c_33));
    CUDA_CHECK(hipFree(d_a_34));
    CUDA_CHECK(hipFree(d_b_34));
    CUDA_CHECK(hipFree(d_c_34));
    CUDA_CHECK(hipFree(d_a_35));
    CUDA_CHECK(hipFree(d_b_35));
    CUDA_CHECK(hipFree(d_c_35));
    CUDA_CHECK(hipFree(d_a_36));
    CUDA_CHECK(hipFree(d_b_36));
    CUDA_CHECK(hipFree(d_c_36));
    CUDA_CHECK(hipFree(d_a_37));
    CUDA_CHECK(hipFree(d_b_37));
    CUDA_CHECK(hipFree(d_c_37));
    CUDA_CHECK(hipFree(d_a_38));
    CUDA_CHECK(hipFree(d_b_38));
    CUDA_CHECK(hipFree(d_c_38));
    CUDA_CHECK(hipFree(d_a_39));
    CUDA_CHECK(hipFree(d_b_39));
    CUDA_CHECK(hipFree(d_c_39));
    CUDA_CHECK(hipFree(d_a_40));
    CUDA_CHECK(hipFree(d_b_40));
    CUDA_CHECK(hipFree(d_c_40));
    CUDA_CHECK(hipFree(d_a_41));
    CUDA_CHECK(hipFree(d_b_41));
    CUDA_CHECK(hipFree(d_c_41));
    CUDA_CHECK(hipFree(d_a_42));
    CUDA_CHECK(hipFree(d_b_42));
    CUDA_CHECK(hipFree(d_c_42));
    CUDA_CHECK(hipFree(d_a_43));
    CUDA_CHECK(hipFree(d_b_43));
    CUDA_CHECK(hipFree(d_c_43));
    CUDA_CHECK(hipFree(d_a_44));
    CUDA_CHECK(hipFree(d_b_44));
    CUDA_CHECK(hipFree(d_c_44));
    CUDA_CHECK(hipFree(d_a_45));
    CUDA_CHECK(hipFree(d_b_45));
    CUDA_CHECK(hipFree(d_c_45));
    CUDA_CHECK(hipFree(d_a_46));
    CUDA_CHECK(hipFree(d_b_46));
    CUDA_CHECK(hipFree(d_c_46));
    CUDA_CHECK(hipFree(d_a_47));
    CUDA_CHECK(hipFree(d_b_47));
    CUDA_CHECK(hipFree(d_c_47));
    CUDA_CHECK(hipFree(d_a_48));
    CUDA_CHECK(hipFree(d_b_48));
    CUDA_CHECK(hipFree(d_c_48));
    CUDA_CHECK(hipFree(d_a_49));
    CUDA_CHECK(hipFree(d_b_49));
    CUDA_CHECK(hipFree(d_c_49));
    CUDA_CHECK(hipFree(d_a_50));
    CUDA_CHECK(hipFree(d_b_50));
    CUDA_CHECK(hipFree(d_c_50));
    CUDA_CHECK(hipFree(d_a_51));
    CUDA_CHECK(hipFree(d_b_51));
    CUDA_CHECK(hipFree(d_c_51));
    CUDA_CHECK(hipFree(d_a_52));
    CUDA_CHECK(hipFree(d_b_52));
    CUDA_CHECK(hipFree(d_c_52));
    CUDA_CHECK(hipFree(d_a_53));
    CUDA_CHECK(hipFree(d_b_53));
    CUDA_CHECK(hipFree(d_c_53));
    CUDA_CHECK(hipFree(d_a_54));
    CUDA_CHECK(hipFree(d_b_54));
    CUDA_CHECK(hipFree(d_c_54));
    CUDA_CHECK(hipFree(d_a_55));
    CUDA_CHECK(hipFree(d_b_55));
    CUDA_CHECK(hipFree(d_c_55));
    CUDA_CHECK(hipFree(d_a_56));
    CUDA_CHECK(hipFree(d_b_56));
    CUDA_CHECK(hipFree(d_c_56));
    CUDA_CHECK(hipFree(d_a_57));
    CUDA_CHECK(hipFree(d_b_57));
    CUDA_CHECK(hipFree(d_c_57));
    CUDA_CHECK(hipFree(d_a_58));
    CUDA_CHECK(hipFree(d_b_58));
    CUDA_CHECK(hipFree(d_c_58));
    CUDA_CHECK(hipFree(d_a_59));
    CUDA_CHECK(hipFree(d_b_59));
    CUDA_CHECK(hipFree(d_c_59));
    CUDA_CHECK(hipFree(d_a_60));
    CUDA_CHECK(hipFree(d_b_60));
    CUDA_CHECK(hipFree(d_c_60));
    CUDA_CHECK(hipFree(d_a_61));
    CUDA_CHECK(hipFree(d_b_61));
    CUDA_CHECK(hipFree(d_c_61));
    CUDA_CHECK(hipFree(d_a_62));
    CUDA_CHECK(hipFree(d_b_62));
    CUDA_CHECK(hipFree(d_c_62));
    CUDA_CHECK(hipFree(d_a_63));
    CUDA_CHECK(hipFree(d_b_63));
    CUDA_CHECK(hipFree(d_c_63));
    CUDA_CHECK(hipFree(d_a_64));
    CUDA_CHECK(hipFree(d_b_64));
    CUDA_CHECK(hipFree(d_c_64));
    CUDA_CHECK(hipFree(d_a_65));
    CUDA_CHECK(hipFree(d_b_65));
    CUDA_CHECK(hipFree(d_c_65));
    CUDA_CHECK(hipFree(d_a_66));
    CUDA_CHECK(hipFree(d_b_66));
    CUDA_CHECK(hipFree(d_c_66));
    CUDA_CHECK(hipFree(d_a_67));
    CUDA_CHECK(hipFree(d_b_67));
    CUDA_CHECK(hipFree(d_c_67));
    CUDA_CHECK(hipFree(d_a_68));
    CUDA_CHECK(hipFree(d_b_68));
    CUDA_CHECK(hipFree(d_c_68));
    CUDA_CHECK(hipFree(d_a_69));
    CUDA_CHECK(hipFree(d_b_69));
    CUDA_CHECK(hipFree(d_c_69));
    CUDA_CHECK(hipFree(d_a_70));
    CUDA_CHECK(hipFree(d_b_70));
    CUDA_CHECK(hipFree(d_c_70));
    CUDA_CHECK(hipFree(d_a_71));
    CUDA_CHECK(hipFree(d_b_71));
    CUDA_CHECK(hipFree(d_c_71));
    CUDA_CHECK(hipFree(d_a_72));
    CUDA_CHECK(hipFree(d_b_72));
    CUDA_CHECK(hipFree(d_c_72));
    CUDA_CHECK(hipFree(d_a_73));
    CUDA_CHECK(hipFree(d_b_73));
    CUDA_CHECK(hipFree(d_c_73));
    CUDA_CHECK(hipFree(d_a_74));
    CUDA_CHECK(hipFree(d_b_74));
    CUDA_CHECK(hipFree(d_c_74));
    CUDA_CHECK(hipFree(d_a_75));
    CUDA_CHECK(hipFree(d_b_75));
    CUDA_CHECK(hipFree(d_c_75));
    CUDA_CHECK(hipFree(d_a_76));
    CUDA_CHECK(hipFree(d_b_76));
    CUDA_CHECK(hipFree(d_c_76));
    CUDA_CHECK(hipFree(d_a_77));
    CUDA_CHECK(hipFree(d_b_77));
    CUDA_CHECK(hipFree(d_c_77));
    CUDA_CHECK(hipFree(d_a_78));
    CUDA_CHECK(hipFree(d_b_78));
    CUDA_CHECK(hipFree(d_c_78));
    CUDA_CHECK(hipFree(d_a_79));
    CUDA_CHECK(hipFree(d_b_79));
    CUDA_CHECK(hipFree(d_c_79));
    CUDA_CHECK(hipFree(d_a_80));
    CUDA_CHECK(hipFree(d_b_80));
    CUDA_CHECK(hipFree(d_c_80));
    CUDA_CHECK(hipFree(d_a_81));
    CUDA_CHECK(hipFree(d_b_81));
    CUDA_CHECK(hipFree(d_c_81));

    CUBLAS_CHECK(hipblasDestroy(cublasH1));CUBLAS_CHECK(hipblasDestroy(cublasH2));CUBLAS_CHECK(hipblasDestroy(cublasH3));CUBLAS_CHECK(hipblasDestroy(cublasH4));CUBLAS_CHECK(hipblasDestroy(cublasH5));CUBLAS_CHECK(hipblasDestroy(cublasH6));CUBLAS_CHECK(hipblasDestroy(cublasH7));CUBLAS_CHECK(hipblasDestroy(cublasH8));CUBLAS_CHECK(hipblasDestroy(cublasH9));CUBLAS_CHECK(hipblasDestroy(cublasH10));CUBLAS_CHECK(hipblasDestroy(cublasH11));CUBLAS_CHECK(hipblasDestroy(cublasH12));CUBLAS_CHECK(hipblasDestroy(cublasH13));CUBLAS_CHECK(hipblasDestroy(cublasH14));CUBLAS_CHECK(hipblasDestroy(cublasH15));CUBLAS_CHECK(hipblasDestroy(cublasH16));CUBLAS_CHECK(hipblasDestroy(cublasH17));CUBLAS_CHECK(hipblasDestroy(cublasH18));CUBLAS_CHECK(hipblasDestroy(cublasH19));CUBLAS_CHECK(hipblasDestroy(cublasH20));CUBLAS_CHECK(hipblasDestroy(cublasH21));CUBLAS_CHECK(hipblasDestroy(cublasH22));CUBLAS_CHECK(hipblasDestroy(cublasH23));CUBLAS_CHECK(hipblasDestroy(cublasH24));CUBLAS_CHECK(hipblasDestroy(cublasH25));CUBLAS_CHECK(hipblasDestroy(cublasH26));CUBLAS_CHECK(hipblasDestroy(cublasH27));CUBLAS_CHECK(hipblasDestroy(cublasH28));CUBLAS_CHECK(hipblasDestroy(cublasH29));CUBLAS_CHECK(hipblasDestroy(cublasH30));CUBLAS_CHECK(hipblasDestroy(cublasH31));CUBLAS_CHECK(hipblasDestroy(cublasH32));CUBLAS_CHECK(hipblasDestroy(cublasH33));CUBLAS_CHECK(hipblasDestroy(cublasH34));CUBLAS_CHECK(hipblasDestroy(cublasH35));CUBLAS_CHECK(hipblasDestroy(cublasH36));CUBLAS_CHECK(hipblasDestroy(cublasH37));CUBLAS_CHECK(hipblasDestroy(cublasH38));CUBLAS_CHECK(hipblasDestroy(cublasH39));CUBLAS_CHECK(hipblasDestroy(cublasH40));CUBLAS_CHECK(hipblasDestroy(cublasH41));CUBLAS_CHECK(hipblasDestroy(cublasH42));CUBLAS_CHECK(hipblasDestroy(cublasH43));CUBLAS_CHECK(hipblasDestroy(cublasH44));CUBLAS_CHECK(hipblasDestroy(cublasH45));CUBLAS_CHECK(hipblasDestroy(cublasH46));CUBLAS_CHECK(hipblasDestroy(cublasH47));CUBLAS_CHECK(hipblasDestroy(cublasH48));CUBLAS_CHECK(hipblasDestroy(cublasH49));CUBLAS_CHECK(hipblasDestroy(cublasH50));CUBLAS_CHECK(hipblasDestroy(cublasH51));CUBLAS_CHECK(hipblasDestroy(cublasH52));CUBLAS_CHECK(hipblasDestroy(cublasH53));CUBLAS_CHECK(hipblasDestroy(cublasH54));CUBLAS_CHECK(hipblasDestroy(cublasH55));CUBLAS_CHECK(hipblasDestroy(cublasH56));CUBLAS_CHECK(hipblasDestroy(cublasH57));CUBLAS_CHECK(hipblasDestroy(cublasH58));CUBLAS_CHECK(hipblasDestroy(cublasH59));CUBLAS_CHECK(hipblasDestroy(cublasH60));CUBLAS_CHECK(hipblasDestroy(cublasH61));CUBLAS_CHECK(hipblasDestroy(cublasH62));CUBLAS_CHECK(hipblasDestroy(cublasH63));CUBLAS_CHECK(hipblasDestroy(cublasH64));CUBLAS_CHECK(hipblasDestroy(cublasH65));CUBLAS_CHECK(hipblasDestroy(cublasH66));CUBLAS_CHECK(hipblasDestroy(cublasH67));CUBLAS_CHECK(hipblasDestroy(cublasH68));CUBLAS_CHECK(hipblasDestroy(cublasH69));CUBLAS_CHECK(hipblasDestroy(cublasH70));CUBLAS_CHECK(hipblasDestroy(cublasH71));CUBLAS_CHECK(hipblasDestroy(cublasH72));CUBLAS_CHECK(hipblasDestroy(cublasH73));CUBLAS_CHECK(hipblasDestroy(cublasH74));CUBLAS_CHECK(hipblasDestroy(cublasH75));CUBLAS_CHECK(hipblasDestroy(cublasH76));CUBLAS_CHECK(hipblasDestroy(cublasH77));CUBLAS_CHECK(hipblasDestroy(cublasH78));CUBLAS_CHECK(hipblasDestroy(cublasH79));CUBLAS_CHECK(hipblasDestroy(cublasH80));CUBLAS_CHECK(hipblasDestroy(cublasH81));

    CUDA_CHECK(hipStreamDestroy(stream1));CUDA_CHECK(hipStreamDestroy(stream2));CUDA_CHECK(hipStreamDestroy(stream3));CUDA_CHECK(hipStreamDestroy(stream4));CUDA_CHECK(hipStreamDestroy(stream5));CUDA_CHECK(hipStreamDestroy(stream6));CUDA_CHECK(hipStreamDestroy(stream7));CUDA_CHECK(hipStreamDestroy(stream8));CUDA_CHECK(hipStreamDestroy(stream9));CUDA_CHECK(hipStreamDestroy(stream10));CUDA_CHECK(hipStreamDestroy(stream11));CUDA_CHECK(hipStreamDestroy(stream12));CUDA_CHECK(hipStreamDestroy(stream13));CUDA_CHECK(hipStreamDestroy(stream14));CUDA_CHECK(hipStreamDestroy(stream15));CUDA_CHECK(hipStreamDestroy(stream16));CUDA_CHECK(hipStreamDestroy(stream17));CUDA_CHECK(hipStreamDestroy(stream18));CUDA_CHECK(hipStreamDestroy(stream19));CUDA_CHECK(hipStreamDestroy(stream20));CUDA_CHECK(hipStreamDestroy(stream21));CUDA_CHECK(hipStreamDestroy(stream22));CUDA_CHECK(hipStreamDestroy(stream23));CUDA_CHECK(hipStreamDestroy(stream24));CUDA_CHECK(hipStreamDestroy(stream25));CUDA_CHECK(hipStreamDestroy(stream26));CUDA_CHECK(hipStreamDestroy(stream27));CUDA_CHECK(hipStreamDestroy(stream28));CUDA_CHECK(hipStreamDestroy(stream29));CUDA_CHECK(hipStreamDestroy(stream30));CUDA_CHECK(hipStreamDestroy(stream31));CUDA_CHECK(hipStreamDestroy(stream32));CUDA_CHECK(hipStreamDestroy(stream33));CUDA_CHECK(hipStreamDestroy(stream34));CUDA_CHECK(hipStreamDestroy(stream35));CUDA_CHECK(hipStreamDestroy(stream36));CUDA_CHECK(hipStreamDestroy(stream37));CUDA_CHECK(hipStreamDestroy(stream38));CUDA_CHECK(hipStreamDestroy(stream39));CUDA_CHECK(hipStreamDestroy(stream40));CUDA_CHECK(hipStreamDestroy(stream41));CUDA_CHECK(hipStreamDestroy(stream42));CUDA_CHECK(hipStreamDestroy(stream43));CUDA_CHECK(hipStreamDestroy(stream44));CUDA_CHECK(hipStreamDestroy(stream45));CUDA_CHECK(hipStreamDestroy(stream46));CUDA_CHECK(hipStreamDestroy(stream47));CUDA_CHECK(hipStreamDestroy(stream48));CUDA_CHECK(hipStreamDestroy(stream49));CUDA_CHECK(hipStreamDestroy(stream50));CUDA_CHECK(hipStreamDestroy(stream51));CUDA_CHECK(hipStreamDestroy(stream52));CUDA_CHECK(hipStreamDestroy(stream53));CUDA_CHECK(hipStreamDestroy(stream54));CUDA_CHECK(hipStreamDestroy(stream55));CUDA_CHECK(hipStreamDestroy(stream56));CUDA_CHECK(hipStreamDestroy(stream57));CUDA_CHECK(hipStreamDestroy(stream58));CUDA_CHECK(hipStreamDestroy(stream59));CUDA_CHECK(hipStreamDestroy(stream60));CUDA_CHECK(hipStreamDestroy(stream61));CUDA_CHECK(hipStreamDestroy(stream62));CUDA_CHECK(hipStreamDestroy(stream63));CUDA_CHECK(hipStreamDestroy(stream64));CUDA_CHECK(hipStreamDestroy(stream65));CUDA_CHECK(hipStreamDestroy(stream66));CUDA_CHECK(hipStreamDestroy(stream67));CUDA_CHECK(hipStreamDestroy(stream68));CUDA_CHECK(hipStreamDestroy(stream69));CUDA_CHECK(hipStreamDestroy(stream70));CUDA_CHECK(hipStreamDestroy(stream71));CUDA_CHECK(hipStreamDestroy(stream72));CUDA_CHECK(hipStreamDestroy(stream73));CUDA_CHECK(hipStreamDestroy(stream74));CUDA_CHECK(hipStreamDestroy(stream75));CUDA_CHECK(hipStreamDestroy(stream76));CUDA_CHECK(hipStreamDestroy(stream77));CUDA_CHECK(hipStreamDestroy(stream78));CUDA_CHECK(hipStreamDestroy(stream79));CUDA_CHECK(hipStreamDestroy(stream80));CUDA_CHECK(hipStreamDestroy(stream81));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}


/**/