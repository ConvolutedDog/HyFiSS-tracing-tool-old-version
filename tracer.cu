#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <stdint.h>
#include <unistd.h>
#include <sys/stat.h>
#include <string>
#include <map>
#include <list>
#include <bitset>
#include <tuple>
#include <vector>
#include <unordered_set>
#include <fstream>
#include <algorithm>
#include <sstream>

#include "utils/utils.h"
#include "nvbit_tool.h"
#include "nvbit.h"
#include "utils/channel.hpp"
#include "common.h"

using namespace std;

#define MAX_KERNELS 300

/* channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
* flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
 pthread_mutex_t mutex;

/* opcode to id map and reverse map  */
map<string, int> opcode_to_id_map;
map<int, string> id_to_opcode_map;


typedef unordered_map<int, int> int_int_map;

// int_int_map bb_map;
int kernel_id = 1;
int bb_id = 0;
int current_sm_id;
int current_cta_id_x;
int current_cta_id_y;
int current_cta_id_z;
int current_warp_id;
int first_warp_exec = 0;
int first_kernel_mem_clk = 0;
static ofstream app_config_fp;
static ofstream insts_trace_fp;
static ofstream issue_config_fp; // yangjianchao16 add
static ofstream instn_config_fp; // yangjianchao16 add
int_int_map reg_dependency_map;
int_int_map pred_dependency_map;
int inst_count = 0;

int kernel_gridX = 0;
int kernel_gridY = 0;
int kernel_gridZ = 0;


typedef map<int, vector<tuple<int, int, int, int>>> SMid_CTAid_Map_t;

/* This map intends to record the SM id that every CTA is issued to during 
 * the exection of one kernel. The key and value of this map:
 * key: CTA id, value: <kernel_id, ctaid.x, ctaid.y, ctaid.z>
 */
SMid_CTAid_Map_t SMid_CTAid_Map;

typedef map<int, vector<uint64_t>> SMid_CTAid_timestamp_Map_t;

SMid_CTAid_timestamp_Map_t SMid_CTAid_timestamp_Map;


// typedef map<int, list<tuple<int, int, int, int, int>>> PCid_CTAid_Map_t;

/* This map intends to record the pc that every instn actually executed on 
 * the fly belongs to during the exection of one kernel. The key and value 
 * of this map:
 * key: <kernel_id, pc, ctaid.x, ctaid.y, ctaid.z, warp_id>, value: string
 */
// PCid_CTAid_Map_t PCid_CTAid_Map;


typedef map<tuple<int, int>, string> PCid_Instn_Map_t;
/* This map intends to record the instruction string that actually executed 
 * on the fly belongs to during the exection of one kernel. The key and value 
 * of this map:
 * key: <kernel_id, pc>, value: string
 */
PCid_Instn_Map_t PCid_Instn_Map;

map<int, int> max_pc;

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We
 * typically do initializations in this call. In this case for instance we get
 * some environment variables values which we use as input arguments to the tool
 */
void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    string pad(100, '-');
    printf("%s\n", pad.c_str());
    
    if (mkdir("configs", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) == -1){
        if( errno == EEXIST ) {
            // alredy exists
            system("rm configs/*");
        } else {
            // something else
            cout << "cannot create configs directory error:" << strerror(errno) << endl;
            throw runtime_error(strerror(errno));
            return;
        }
    }

    if (mkdir("memory_traces", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) == -1){
        if( errno == EEXIST ) {
        // alredy exists
        system("rm memory_traces/*");
        } else {
        // something else
            cout << "cannot create memory_traces directory error:" << strerror(errno) << endl;
            throw runtime_error(strerror(errno));
            return;
        }
    }

    if (mkdir("sass_traces", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) == -1){
        if( errno == EEXIST ) {
        // alredy exists
        system("rm sass_traces/*");
        } else {
        // something else
            cout << "cannot create sass_traces directory error:" << strerror(errno) << endl;
            throw runtime_error(strerror(errno));
            return;
        }
    }

    app_config_fp.open("./configs/app.config");
    app_config_fp << "####################################################################################\n";
    app_config_fp << "######                                                                        ######\n";
    app_config_fp << "######                 The app.config file for the simulator.                 ######\n";
    app_config_fp << "######                                                                        ######\n";
    app_config_fp << "####################################################################################\n\n";

    issue_config_fp.open("./configs/issue.config");
    issue_config_fp << "####################################################################################\n";
    issue_config_fp << "######                                                                        ######\n";
    issue_config_fp << "######                The issue.config file for the simulator.                ######\n";
    issue_config_fp << "######                                                                        ######\n";
    issue_config_fp << "####################################################################################\n";
    issue_config_fp << "\n";
    issue_config_fp << "# trace_issued_sms_num : the number of SMs that have issued at least one warp.\n";
    issue_config_fp << "# trace_issued_sm_id_x ctas_num : the number of CTAs that have been issued on SM x.\n";
    issue_config_fp << "# trace_issued_sm_id_x tuple_list : the list of CTAs that have been issued on SM x,\n";
    issue_config_fp << "#                                   format (kernel_id, cta.x, cta.y, cta.z)\n";
    issue_config_fp << "# trace_issued_sm_id_x is the list of issued CTAs on SM x.\n";
    issue_config_fp << "# Note that the list of issued CTAs on SM x, does not represent the order in which\n";
    issue_config_fp << "# CTAs are issued.\n\n";

    instn_config_fp.open("./configs/instn.config");
    instn_config_fp << "####################################################################################\n";
    instn_config_fp << "######                                                                        ######\n";
    instn_config_fp << "######                The instn.config file for the simulator.                ######\n";
    instn_config_fp << "######                                                                        ######\n";
    instn_config_fp << "####################################################################################\n\n";
}


void dump_app_config(){
    app_config_fp << "-app_kernels_id ";
    for (int i = 1; i < kernel_id; i++){
        if (i > 1) {
            app_config_fp << ",";
        }
        app_config_fp << i;
    }
    app_config_fp.close();
    cout << "--> sass + memory traces are collected for "<<(kernel_id - 1) << " kernels"<<"\n";
}

/* set used to avoid re-instrumenting the same functions multiple times */
unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* get related functions of the kernel (device function that can be
     * called by the kernel) */
    vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);
         

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        // /* get the static control flow graph of instruction */
        // const CFG_t &cfg = nvbit_get_CFG(ctx, f);
        // if (cfg.is_degenerate) {
        //     printf("Warning: Function %s is degenerated, we can't compute basic "
        //         "blocks statically",
        //         nvbit_get_func_name(ctx, f));
        // }
        // int bb_id = 0;
        // int count = 0;
        // cout<<"--->kernel"<<kernel_id<<"\n";
        // /* iterate on basic block and inject the first instruction */
        // for (auto &bb : cfg.bbs) {
        //      cout <<"BB"<<count<<"\n";
        //      for (auto &i : bb->instrs) {
        //          i->print(" ");
        //      }
        //     bb_id++;
        // }

        const vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

        uint32_t cnt = 0;
        /* iterate on the static instructions */
        for (auto instr : instrs) {
            if (cnt < instr_begin_interval || cnt >= instr_end_interval) {
                cnt++;
                continue;
            }

            if (opcode_to_id_map.find(instr->getOpcode()) == opcode_to_id_map.end()) {
                int opcode_id = opcode_to_id_map.size();
                opcode_to_id_map[instr->getOpcode()] = opcode_id;
                id_to_opcode_map[opcode_id] = string(instr->getOpcode());
            }
            int opcode_id = opcode_to_id_map[instr->getOpcode()];
            int is_glob_loc = 0;
            int pred_num = -1;
            int mref_id = 0;
            int dst_oprnd = -1;
            int dst_oprnd_type = -1;
            int src_oprnds[5] = {-1};
            int src_oprnds_type[5] = {-1};     
            /*
            operands types:
                1 = REG & UREG
                2 = PRED & UPRED
                3 = MREF
            // ignore immediate and CBANK since it will not affect
            // the dependency check when printing out the instruction
            // predciates are also resolved at runtime, so we don't care much about but leave it for now
            */
            

            /* for cache memories */
            if( instr->getMemorySpace() == InstrType::MemorySpace::GLOBAL ||
                instr->getMemorySpace() == InstrType::MemorySpace::LOCAL  ||
                instr->getMemorySpace() == InstrType::MemorySpace::GENERIC  ){
                is_glob_loc = 1;
            }

            if(instr->hasPred()){
                pred_num = (int)instr->getPredNum();
            }

            

            /* insert call to the instrumentation function with its arguments */
            nvbit_insert_call(instr, "instrument_inst", IPOINT_BEFORE);
            /* predicate value */
            nvbit_add_call_arg_guard_pred_val(instr);
            /* programm counter */
            nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());
            /* opcode id */
            nvbit_add_call_arg_const_val32(instr, opcode_id);
            /* global or local mem. instruction? */
            nvbit_add_call_arg_const_val32(instr, is_glob_loc);
            /* memory reference 64 bit address */
            nvbit_add_call_arg_mref_addr64(instr, mref_id);

            for (int i = 0; i < instr->getNumOperands(); i++) {
                const InstrType::operand_t *op = instr->getOperand(i);
                
                if (i == 0){ //handle dest oprnd
                    if(op->type == InstrType::OperandType::REG 
                        || op->type == InstrType::OperandType::UREG){ //dest oprnd is register 
                        dst_oprnd = op->u.reg.num;
                        dst_oprnd_type = 1;
                    } else if (op->type == InstrType::OperandType::PRED
                        || op->type == InstrType::OperandType::UPRED){ //1 oprnd is const immediate UINT64
                        dst_oprnd = op->u.pred.num;
                        dst_oprnd_type = 2;
                    } else if (op->type == InstrType::OperandType::MREF){ //dest oprnd is memory (i.e.: ST or REG)
                        if (is_glob_loc){
                            mref_id ++;
                        }
                        dst_oprnd_type = 3;
                        if(op->u.mref.has_ra){
                            // e.g., STG.E.64.SYS [R8], R6; [R8]  is Return Address Register
                            //       STS.64 [R69], R58 ;    [R69] is Return Address Register
                            dst_oprnd = op->u.mref.ra_num;
                        }else if(op->u.mref.has_ur){
                            // may unified addressing mode, rarely encountered such a situation
                            dst_oprnd = op->u.mref.ur_num;
                        }
                    }
                }else{ //handle src oprnds
                    if(op->type == InstrType::OperandType::REG 
                        || op->type == InstrType::OperandType::UREG){
                        src_oprnds[i] = op->u.reg.num;
                        src_oprnds_type[i] = 1;
                    } else if (op->type == InstrType::OperandType::PRED
                        || op->type == InstrType::OperandType::UPRED){
                        src_oprnds[i] = op->u.reg.num;
                        src_oprnds_type[i] = 2;
                    } else if (op->type == InstrType::OperandType::MREF){
                        if (is_glob_loc){
                            mref_id ++;
                        }
                        src_oprnds_type[i] = 3;
                        if(op->u.mref.has_ra){
                            src_oprnds[i] = op->u.mref.ra_num;
                        }else if(op->u.mref.has_ur){
                            src_oprnds[i] = op->u.mref.ur_num;
                        }
                    }
                }
            }

            /* memory references */
            nvbit_add_call_arg_const_val32(instr, mref_id);
            /* handle LDGSTS instruction with 2 memory references */
            if(mref_id == 2){
                nvbit_add_call_arg_mref_addr64(instr, 1);
            }else{
                nvbit_add_call_arg_mref_addr64(instr, 0);
            }

            /* destination operand */
            nvbit_add_call_arg_const_val32(instr, dst_oprnd);
            nvbit_add_call_arg_const_val32(instr, dst_oprnd_type);

            /* source operands */
            nvbit_add_call_arg_const_val32(instr, src_oprnds[0]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds_type[0]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds[1]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds_type[1]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds[2]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds_type[2]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds[3]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds_type[3]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds[4]);
            nvbit_add_call_arg_const_val32(instr, src_oprnds_type[4]);

            /* predicate num */
            nvbit_add_call_arg_const_val32(instr, pred_num);

            nvbit_add_call_arg_const_val32(instr, (int)(instr->isPredNeg()));
            nvbit_add_call_arg_const_val32(instr, (int)(instr->isPredUniform()));

            /* add pointer to channel_dev*/
            nvbit_add_call_arg_const_val64(instr, (uint64_t)&channel_dev);

            cnt++; 
        }
    }
}

#include <algorithm>
#include <iterator>

void dump_issue_config(){
    vector<int> issued_sms_num;

    issue_config_fp << "-trace_issued_sms_num " << SMid_CTAid_Map.size() << "\n";
    
    int idx1 = 0;
    for (auto it_map_issue = SMid_CTAid_Map.begin(); it_map_issue != SMid_CTAid_Map.end(); it_map_issue++) {
        
        if (issued_sms_num.size() < 80) {// TO DO: replace 70 by confiure
            auto iter = find(issued_sms_num.begin(), issued_sms_num.end(), it_map_issue->first);
            /* Don't find it_map_issue->first in issued_sms_num, need to add it. */
            if (iter == issued_sms_num.end()) {
                issued_sms_num.push_back(it_map_issue->first);
            }
        }
        
        issue_config_fp << "-trace_issued_sm_id_" << it_map_issue->first << " " 
                        << it_map_issue->second.size() << ","
                        << it_map_issue->first << ",";
        
        // cout << it_map_issue->first << " " << SMid_CTAid_timestamp_Map[it_map_issue->first].size() << endl;
         
        int idx2 = 0;
        for (auto it_tuple = it_map_issue->second.begin(); it_tuple != it_map_issue->second.end(); it_tuple++) {

            issue_config_fp << "(" << get<0>(*it_tuple) << ",";
            // issue_config_fp << get<1>(*it_tuple) << ",";
            // issue_config_fp << get<2>(*it_tuple) << ",";
            // issue_config_fp << get<3>(*it_tuple) << "),";
            /* calculate an index for the block the current instn belong to */
            int index = get<3>(*it_tuple) * kernel_gridY * kernel_gridX + kernel_gridX * get<2>(*it_tuple)  + get<1>(*it_tuple);

            // cout << "    " << it_map_issue->first << " " << idx2 << endl;
            uint64_t timestamp = SMid_CTAid_timestamp_Map[it_map_issue->first][idx2];

            issue_config_fp << index 
                            << "," << hex << timestamp << dec// ????????????????????????????
                            << "),";
            idx2++;
        }
        issue_config_fp << "\n";
        idx1++;
    }

    issue_config_fp << "-trace_issued_sms_vector "; 
    for (auto sm_num : issued_sms_num)
        issue_config_fp << sm_num << ",";
    issue_config_fp << endl;

    issue_config_fp.close();
}


void dump_instn_config(){
    for (int kernel = 1; kernel < kernel_id; kernel++){
        for (int pc = 0; pc < 10000000; pc += 16) {
            bool found = false;
            for (const auto& item : PCid_Instn_Map) {
                if (get<0>(item.first) == kernel && get<1>(item.first) == pc) {
                    instn_config_fp << item.second << endl;
                    found = true;
                }
            }
            if (!found) {
                break;
            }
        }
    }
    instn_config_fp.close();
}

__global__ void flush_channel() {
    /* push memory access with negative cta id to communicate the kernel is completed */
    inst_access_t ma;
    ma.cta_id_x = -1;
    channel_dev.push(&ma, sizeof(inst_access_t));
    // /* flush channel */
    channel_dev.flush();
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                        const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag) return;

    if (kernel_id > MAX_KERNELS){
        exit(0);
    } 

    if (cbid == API_CUDA_cuLaunchKernel_ptsz || cbid == API_CUDA_cuLaunchKernel) {

        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

            if (!is_exit) {
                pthread_mutex_lock(&mutex);
                instrument_function_if_needed(ctx, p->f);
                nvbit_enable_instrumented(ctx, p->f, true);
                recv_thread_receiving = true;

                cout << "Starting kernel #" << kernel_id << "...\n";

                kernel_gridX = p->gridDimX;
                kernel_gridY = p->gridDimY;
                kernel_gridZ = p->gridDimZ;

                string file_name = "./sass_traces/kernel_"+ to_string(kernel_id) + ".sass";
                insts_trace_fp.open(file_name);
            } else {
                /* make sure current kernel is completed */
                hipDeviceSynchronize();
                assert(hipGetLastError() == hipSuccess);

                /* make sure we prevent re-entry on the nvbit_callback when issuing
                * the flush_channel kernel */
                skip_flag = true;

                /* issue flush of channel so we are sure all the accesses
                * have been pushed */
                flush_channel<<<1, 1>>>();
                hipDeviceSynchronize();
                assert(hipGetLastError() == hipSuccess);

                /* unset the skip flag */
                skip_flag = false;
                
                /* wait here until the receiving thread has not finished with the current kernel */
                while (recv_thread_receiving) {
                    pthread_yield();
                }

                int gridX = 0, gridY = 0, gridZ = 0, blockX = 0, blockY = 0, blockZ= 0,\
                nregs=0, shmem_static_nbytes=0, shmem_dynamic_nbytes = 0, stream_id = 0;

                CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));
                CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes,HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

                gridX = p->gridDimX;
                gridY = p->gridDimY;
                gridZ = p->gridDimZ;
                blockX = p->blockDimX;
                blockY = p->blockDimY;
                blockZ = p->blockDimZ;
                stream_id = (uint64_t)p->hStream;
                shmem_dynamic_nbytes = p->sharedMemBytes;

                int num_ctas = gridX * gridY * gridZ;

                int threads_per_cta = blockX * blockY * blockZ;
                int tot_num_thread = num_ctas * threads_per_cta;
                int tot_num_warps =  tot_num_thread/32;
                if(tot_num_warps ==0)
                    tot_num_warps = 1;

                string kernel_name = nvbit_get_func_name(ctx, p->f);
                std::cout << kernel_name << endl;
                string delimiter = "(";
                kernel_name  = kernel_name.substr(0, kernel_name.find(delimiter));
                replace(kernel_name.begin(), kernel_name.end(), ' ', '_');

                app_config_fp << "-kernel_" + to_string(kernel_id) << "_kernel_name " << kernel_name << "\n"; // BUG
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_num_registers " << nregs << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_shared_mem_bytes " 
                              << (shmem_static_nbytes + shmem_dynamic_nbytes) << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_grid_size " << num_ctas << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_block_size " << threads_per_cta << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_cuda_stream_id " << stream_id << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_grid_dim_x " << gridX << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_grid_dim_y " << gridY << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_grid_dim_z " << gridZ << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_tb_dim_x " << blockX << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_tb_dim_y " << blockY << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_tb_dim_z " << blockZ << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_shmem_base_addr " 
                              << (uint64_t)nvbit_get_shmem_base_addr(ctx) << "\n";
                app_config_fp << "-kernel_" + to_string(kernel_id) << "_local_base_addr " 
                              << (uint64_t)nvbit_get_local_mem_base_addr(ctx) << "\n";
                
                cout << "Exiting kernel #" << kernel_id << "...\n";
                kernel_id++;
                first_warp_exec = 0;
                first_kernel_mem_clk = 0;
                inst_count = 0;
                reg_dependency_map.clear();
                pred_dependency_map.clear();
                insts_trace_fp.close();
                
                pthread_mutex_unlock(&mutex);
            }
    }
}

uint64_t kernel_mem_clk = 0;

void *recv_thread_fun(void *) {
    char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

    while (recv_thread_started) {
        uint32_t num_recv_bytes = 0;

        if (recv_thread_receiving && 
            (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) > 0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {

                inst_access_t *ia = (inst_access_t *)&recv_buffer[num_processed_bytes];

                if (ia->cta_id_x == -1) {
                    recv_thread_receiving = false;
                    break;
                }
                if (first_warp_exec == 0){
                    current_sm_id = ia->sm_id;
                    current_cta_id_x = ia->cta_id_x;
                    current_cta_id_y = ia->cta_id_y;
                    current_cta_id_z = ia->cta_id_z;
                    current_warp_id = ia->warp_id;
                    first_warp_exec = 1;
                }

                // if(ia->sm_id == current_sm_id && ia->cta_id_x == current_cta_id_x && ia->cta_id_y == current_cta_id_y && ia->cta_id_z == current_cta_id_z){
                        /* the SM id that every CTA is issued to */
                        auto it_map_smid_ctaid = SMid_CTAid_Map.find(ia->sm_id);                                          // yangjianchao16 add
                        auto it_map_smid_cta_id_timestamp = SMid_CTAid_timestamp_Map.find(ia->sm_id);                     // yangjianchao16 add
                        auto item_tuple = make_tuple(kernel_id, ia->cta_id_x, ia->cta_id_y, ia->cta_id_z);                // yangjianchao16 add
                        if (it_map_smid_ctaid == SMid_CTAid_Map.end()){ // the first time sm_id occurs                    // yangjianchao16 add
                            SMid_CTAid_Map[ia->sm_id].push_back(item_tuple);                                              // yangjianchao16 add
                            SMid_CTAid_timestamp_Map[ia->sm_id].push_back(ia->curr_clk);                                  // yangjianchao16 add
                        } else { // not the first time sm_id occurs                                                       // yangjianchao16 add
                            if (find(it_map_smid_ctaid->second.begin(), it_map_smid_ctaid->second.end(), item_tuple) ==   // yangjianchao16 add
                                it_map_smid_ctaid->second.end()) {                                                        // yangjianchao16 add
                                it_map_smid_ctaid->second.push_back(item_tuple);                                          // yangjianchao16 add
                                it_map_smid_cta_id_timestamp->second.push_back(ia->curr_clk);                             // yangjianchao16 add
                            }                                                                                             // yangjianchao16 add
                        }                                                                                                 // yangjianchao16 add

                        /* the pc that every instn actually executed on the fly belongs to */
                        auto it_map_pcid_instn = PCid_Instn_Map.find(make_tuple(kernel_id, ia->pc));                      // yangjianchao16 add
                        if (it_map_pcid_instn == PCid_Instn_Map.end()){ // the first time <kernel_id, pc> occurs          // yangjianchao16 add
                            string Instn_string = "";                                                                     // yangjianchao16 add
                            /* kernel_id, pc */
                            stringstream ss;                                                                         // yangjianchao16 add
                            ss << hex << ia->pc;                                                                          // yangjianchao16 add
                            Instn_string += to_string(kernel_id) + " " + ss.str() + " ";                                  // yangjianchao16 add
                            /* pred */
                            if (ia->pred_num != -1) {                                                                     // yangjianchao16 add
                                if (ia->isPredNeg)                                                                        // yangjianchao16 add
                                    Instn_string += "@!P" + to_string(ia->pred_num) + " ";                                // yangjianchao16 add
                                else                                                                                      // yangjianchao16 add
                                    Instn_string += "@P" + to_string(ia->pred_num) + " ";                                 // yangjianchao16 add
                            }                                                                                             // yangjianchao16 add
                            /* opcode */
                            Instn_string += id_to_opcode_map[ia->opcode_id] + " ";                                        // yangjianchao16 add
                            /* destination operands */
                            if (ia->dst_oprnd_type == 1){                                                                 // yangjianchao16 add
                                Instn_string += "R" + to_string(ia->dst_oprnd) + " ";                                     // yangjianchao16 add
                            } else if (ia->dst_oprnd_type == 2){                                                          // yangjianchao16 add
                                Instn_string += "P" + to_string(ia->dst_oprnd) + " ";                                     // yangjianchao16 add
                            } else if (ia->dst_oprnd_type == 3){                                                          // yangjianchao16 add
                                Instn_string += "[R" + to_string(ia->dst_oprnd) + "] ";                                   // yangjianchao16 add
                            }                                                                                             // yangjianchao16 add
                            /* src operands */
                            for (int i = 0; i < 5; i++){                                                                  // yangjianchao16 add
                                if (ia->src_oprnds[i] != -1){                                                             // yangjianchao16 add
                                    if (ia->src_oprnds_type[i] == 1 || ia->src_oprnds_type[i] == 3){                      // yangjianchao16 add
                                        Instn_string += "R" + to_string(ia->src_oprnds[i]) + " ";                         // yangjianchao16 add
                                    } else if (ia->src_oprnds_type[i] == 2){                                              // yangjianchao16 add
                                        Instn_string += "P" + to_string(ia->src_oprnds[i]) + " ";                         // yangjianchao16 add
                                    }                                                                                     // yangjianchao16 add
                                }                                                                                         // yangjianchao16 add
                            }                                                                                             // yangjianchao16 add
                            PCid_Instn_Map[make_tuple(kernel_id, ia->pc)] = Instn_string;                                 // yangjianchao16 add
                        }                                                                                                 // yangjianchao16 add

                        // insts_trace_fp << "PC-0x" << hex << ia->pc << dec << " ";
                        // insts_trace_fp << "CLK-0x" << hex << ia->curr_clk << dec << " ";
                        // insts_trace_fp << "sm_id-" << ia->sm_id << " " 
                        //                << "cta_id_x-" << ia->cta_id_x << " " 
                        //                << "cta_id_y-" << ia->cta_id_y << " " 
                        //                << "cta_id_z-" << ia->cta_id_z << " " 
                        //                << "warp_id-" << ia->warp_id<<" "
                        //                << "gwarp_id-" << ia->gwarp_id<<" ";
                        
                        insts_trace_fp << hex << ia->pc << dec << " " << ia->gwarp_id << " ";

                        // /* opcode */
                        // insts_trace_fp<<id_to_opcode_map[ia->opcode_id]<<" ";

                        // /* destination operands */
                        // if(ia->dst_oprnd_type == 1 || ia->dst_oprnd_type == 3){
                        //     insts_trace_fp<<"R"<<ia->dst_oprnd << " ";
                        // }else if (ia->dst_oprnd_type == 2){
                        //     insts_trace_fp<<"P"<<ia->dst_oprnd << " ";
                        // }
                        
                        // /* src operands */
                        // for(int i=0; i<5; i++){
                        //     if(ia->src_oprnds[i] != -1){
                        //         if (ia->src_oprnds_type[i] == 1 || ia->src_oprnds_type[i] == 3){
                        //             insts_trace_fp<<"R"<<ia->src_oprnds[i]<<" ";
                        //         }else if(ia->src_oprnds_type[i] == 2){
                        //             insts_trace_fp<<"P"<<ia->src_oprnds[i]<< " ";
                        //         }
                        //     }
                        // }

                        // insts_trace_fp<<"\n";
                // }


                // bitset<32> mask(ia->active_mask & ia->predicate_mask);
                if (ia->is_mem_inst == 1){
                    if (first_kernel_mem_clk == 0) {
                        kernel_mem_clk = ia->curr_clk;
                        first_kernel_mem_clk = 1;
                    }

                    ofstream mem_trace_fp;

                    /* calculate an index for the block the current mem reference belong to */
                    int index = ia->cta_id_z * kernel_gridY * kernel_gridX + kernel_gridX * ia->cta_id_y  + ia->cta_id_x;

                    string file_name = "./memory_traces/kernel_"+ to_string(kernel_id) + "_block_"+to_string(index)+".mem";
                    mem_trace_fp.open(file_name, ios::app);
                    
                    // mem_trace_fp << "\n=====\n";
                    // mem_trace_fp << id_to_opcode_map[ia->opcode_id] << " ";
                    
                    mem_trace_fp << hex << ia->pc << " ";

                    // mem_trace_fp << id_to_opcode_map[ia->opcode_id] << " ";

                    // mem_trace_fp << hex << (ia->active_mask & ia->predicate_mask) << " ";
                    
                    mem_trace_fp << hex << int(ia->curr_clk) - int(kernel_mem_clk) << " ";
                    // for (int m = 0; m < 32; m++) {
                    //     if(ia->mem_addrs1[m]!=0){
                    //         mem_trace_fp<<"0x"<<hex<<ia->mem_addrs1[m]<<" ";
                    //     } 
                    // }
                    mem_trace_fp << hex << ia->mem_addrs1[0] << " ";
                    // if (ia->mref_id == 2){
                    //     for (int m = 0; m < 32; m++) {
                    //         if(ia->mem_addrs2[m]!=0){
                    //             mem_trace_fp << hex << ia->mem_addrs2[m]<<" ";
                    //         } 
                    //     }
                    // }
                    if (ia->mref_id == 2) mem_trace_fp << hex << ia->mem_addrs2[0] << " ";
                    mem_trace_fp << endl;
                    mem_trace_fp.close();
                }
                
                num_processed_bytes += sizeof(inst_access_t);
            }  
        } 
    }
    free(recv_buffer);
    return NULL;
}


void nvbit_at_ctx_init(hipCtx_t ctx) {
    recv_thread_started = true;
    channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
    pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}


void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (recv_thread_started) {
        recv_thread_started = false;
        pthread_join(recv_thread, NULL);
    }

    dump_app_config();
    dump_issue_config();
    dump_instn_config();
}
